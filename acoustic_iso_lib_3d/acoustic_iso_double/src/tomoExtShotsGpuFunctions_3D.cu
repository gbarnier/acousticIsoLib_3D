#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>
#include "tomoExtShotsGpuFunctions_3D.h"
#include "varDeclare_3D.h"
#include "kernelsGpu_3D.cu"
#include "cudaErrors_3D.cu"
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <ctime>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>

/******************************************************************************/
/****************** Declaration of auxiliary functions ************************/
/******************************************************************************/
// Note: The implementations of these auxiliary functions are done at the bottom of the file
void computeTomoSrcWfldDt2_3D(double *dev_sourcesIn, double *wavefield1, long long *dev_sourcesPositionsRegIn, int nSourcesRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamIn);

void computeTomoRecWfld_3D(double *dev_dataRegDtsIn, double *wavefield2, long long *dev_receiversPositionsRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nblockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamIn);

// Forward + Subsurface offsets
void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nblockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn);

void computeTomoLeg2HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nblockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn);

// Forward + Time-lags
// void computeTomoLeg1TauFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nblockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn);
//
// void computeTomoLeg2TauFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nblockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn);

// Adjoint + Subsurface offsets
void computeTomoLeg1HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn);

void computeTomoLeg2HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn);

// Adjoint + Time-lags
// void computeTomoLeg1TauAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn);
//
// void computeTomoLeg2TauAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn);

/******************************************************************************/
/**************************** Initialization **********************************/
/******************************************************************************/
/* Parameter settings */
bool getGpuInfo_3D(std::vector<int> gpuList, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << gpuList.size() << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;
		std::cout << "Id of requested GPUs: ";
		for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
			if (iGpu<gpuList.size()-1){std::cout << gpuList[iGpu] << ", ";}
 			else{ std::cout << gpuList[iGpu] << std::endl;}
		}

		// Driver version
		std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl;

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

	// Check that the number of requested GPU is less or equal to the total number of available GPUs
	if (gpuList.size()>nDevice) {
		std::cout << "**** ERROR [getGpuInfo_3D]: Number of requested GPU greater than available GPUs ****" << std::endl;
		return false;
	}

	// Check that the GPU numbers in the list are between 0 and nGpu-1
	for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
		if (gpuList[iGpu]<0 || gpuList[iGpu]>nDevice-1){
			std::cout << "**** ERROR [getGpuInfo_3D]: One of the element of the GPU Id list is not a valid GPU Id number ****" << std::endl;
			return false;
		}
	}

	return true;
}

/* Initialize GPU */
void initTomoExtGpu_3D(double dz, double dx, double dy, int nz, int nx, int ny, int nts, double dts, int sub, int minPad, int blockSize, double alphaCos, int nExt1, int nExt2, int leg1, int leg2, int nGpu, int iGpuId, int iGpuAlloc){

	// Set GPU
	hipSetDevice(iGpuId);

	// Host variables
	host_nz = nz;
	host_nx = nx;
    host_ny = ny;
	host_yStride = nz * nx;
	host_nts = nts;
	host_dts = dts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;
    host_nExt1 = nExt1;
    host_nExt2 = nExt2;
	host_hExt1 = (nExt1-1)/2;
    host_hExt2 = (nExt2-1)/2;
    host_nModelExt = nz * nx * ny * nExt1 * nExt2;
    host_nVel = nz * nx * ny;
	host_extStride = host_nExt1 * host_nVel;
	host_leg1 = leg1;
	host_leg2 = leg2;

	// Coefficients for second-order time derivative
	host_cSide = 1.0 / (host_dts*host_dts);
	host_cCenter = -2.0 / (host_dts*host_dts);
	//
	// host_cSide = 0.0;
	// host_cCenter = 1.0;

	/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
	// Only one GPU will perform the following
	if (iGpuId == iGpuAlloc) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		dev_pLeft = new double*[nGpu];
		dev_pRight = new double*[nGpu];
		dev_pTemp = new double*[nGpu];

		dev_pDt0 = new double*[nGpu];
		dev_pDt1 = new double*[nGpu];
		dev_pDt2 = new double*[nGpu];
		dev_pDtTemp = new double*[nGpu];

		dev_pSourceWavefield = new double*[nGpu];
		dev_pRecWavefield = new double*[nGpu];

		// Data and model
		dev_dataRegDts = new double*[nGpu];
		dev_dataRegDtsQc = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new long long*[nGpu];
		dev_receiversPositionReg = new long long*[nGpu];

        // Sources signal
		dev_sourcesSignals = new double*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

        // Reflectivity scaling
		dev_reflectivityScale = new double*[nGpu];

        // Background perturbation ("model" for tomo)
		dev_modelTomo = new double*[nGpu];

		// Extended reflectivity for tomo
		dev_extReflectivity = new double*[nGpu];

		// Debug model and data
		dev_modelDebug = new double*[nGpu];
		dev_dataDebug = new double*[nGpu];

        // Streams
		compStream = new hipStream_t[nGpu];
		transferStream = new hipStream_t[nGpu];
		transferStreamH2D = new hipStream_t[nGpu];
		transferStreamD2H = new hipStream_t[nGpu];
		pin_wavefieldSlice1 = new double*[nGpu];
		pin_wavefieldSlice2 = new double*[nGpu];
		dev_pStream = new double*[nGpu];

		// Time-lags
		dev_pSourceWavefieldTau = new double**[nGpu];
		for (int iGpu=0; iGpu<nGpu; iGpu++){
			dev_pSourceWavefieldTau[iGpu] = new double*[4*host_hExt1+1];
		}
		dev_pTempTau = new double*[nGpu];

	}
	/**************************** COMPUTE LAPLACIAN COEFFICIENTS ************************/
	double host_coeff[COEFF_SIZE] = get_coeffs((double)dz,(double)dx,(double)dy); // Stored on host

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	// Time interpolation filter length / half length
	int hInterpFilter = host_sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Subsampling parameter for time interpolation is too high ****" << std::endl;
		assert (1==2);
	}

	// Allocate and fill time interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Padding value is too high ****" << std::endl;
		assert (1==2);
	}
	double cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		double arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Blocksize value from parfile does not match value from varDeclare file ****" << std::endl;
		assert (1==2);
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_coeff), host_coeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nTimeInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hTimeInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_timeInterpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(double), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ny), &ny, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nVel), &host_nVel, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_yStride), &host_yStride, sizeof(long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nModelExt), &host_nModelExt, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));

	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

    // Extension parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nExt1), &host_nExt1, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nExt2), &host_nExt2, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hExt1), &host_hExt1, sizeof(int), 0, hipMemcpyHostToDevice));
    cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hExt2), &host_hExt2, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_extStride), &host_extStride, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));

	// Second order time derivative coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cCenter), &host_cCenter, sizeof(double), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cSide), &host_cSide, sizeof(double), 0, hipMemcpyHostToDevice));

}

/* Allocation on device */
void allocateTomoExtShotsGpu_3D(double *vel2Dtw2, double *reflectivityScale, double *extReflectivity, int iGpu, int iGpuId){

	// Get GPU number
	hipSetDevice(iGpuId);

	// Scaled velocity
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nVel*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nVel*sizeof(double), hipMemcpyHostToDevice));

    // Reflectivity scale
	cuda_call(hipMalloc((void**) &dev_reflectivityScale[iGpu], host_nVel*sizeof(double))); // Allocate scaling for reflectivity
	cuda_call(hipMemcpy(dev_reflectivityScale[iGpu], reflectivityScale, host_nVel*sizeof(double), hipMemcpyHostToDevice)); //

	// Allocate time slices on device
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nVel*sizeof(double))); // Allocate time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nVel*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_pLeft[iGpu], host_nVel*sizeof(double)));
    cuda_call(hipMalloc((void**) &dev_pRight[iGpu], host_nVel*sizeof(double)));

	// Allocate time slices on device for second time derivative of source wavefield
	cuda_call(hipMalloc((void**) &dev_pDt0[iGpu], host_nVel*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_pDt1[iGpu], host_nVel*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_pDt2[iGpu], host_nVel*sizeof(double)));

    // Reflectivity model
    cuda_call(hipMalloc((void**) &dev_modelTomo[iGpu], host_nVel*sizeof(double)));

	// Allocate pinned memory on host
	cuda_call(hipHostAlloc((void**) &pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipHostMallocDefault));
	cuda_call(hipHostAlloc((void**) &pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipHostMallocDefault));

	// Allocate the slice where we store the wavefield slice before transfering it to the host's pinned memory
	cuda_call(hipMalloc((void**) &dev_pStream[iGpu], host_nVel*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_pSourceWavefield[iGpu], host_nVel*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_pRecWavefield[iGpu], host_nVel*sizeof(double)));

	// Allocate and copy from host to device extended reflectivity
	cuda_call(hipMalloc((void**) &dev_extReflectivity[iGpu], host_nModelExt*sizeof(double)));
	cuda_call(hipMemcpy(dev_extReflectivity[iGpu], extReflectivity, host_nModelExt*sizeof(double), hipMemcpyHostToDevice));

}

/* Deallocation on device */
void deallocateTomoExtShotsGpu_3D(int iGpu, int iGpuId){

	hipSetDevice(iGpuId);
	cuda_call(hipFree(dev_vel2Dtw2[iGpu]));
    cuda_call(hipFree(dev_reflectivityScale[iGpu]));
	cuda_call(hipFree(dev_extReflectivity[iGpu]));
	cuda_call(hipFree(dev_p0[iGpu]));
	cuda_call(hipFree(dev_p1[iGpu]));
    cuda_call(hipFree(dev_pLeft[iGpu]));
    cuda_call(hipFree(dev_pRight[iGpu]));
	cuda_call(hipFree(dev_pDt0[iGpu]));
	cuda_call(hipFree(dev_pDt1[iGpu]));
	cuda_call(hipFree(dev_pDt2[iGpu]));
    cuda_call(hipFree(dev_pSourceWavefield[iGpu]));
	cuda_call(hipFree(dev_pRecWavefield[iGpu]));
	cuda_call(hipFree(dev_modelTomo[iGpu]));
	cuda_call(hipHostFree(pin_wavefieldSlice1[iGpu]));
	cuda_call(hipHostFree(pin_wavefieldSlice2[iGpu]));
}

/******************************************************************************/
/************************* Tomo extended forward ******************************/
/******************************************************************************/
// Subsurface offsets
void tomoHxHyShotsFwdGpu_3D(double *model, double *dataRegDts, double *extReflectivity, double *sourcesSignals, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, double *wavefield1, double *wavefield2, int iGpu, int iGpuId){

	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&compStream[iGpu]);
	hipStreamCreate(&transferStreamH2D[iGpu]);
	hipStreamCreate(&transferStreamD2H[iGpu]);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Sources geometry + signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Blocks for Laplacian
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocks data recording
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	/**************************************************************************/
	/****************************** Source ************************************/
	/**************************************************************************/
	// The wavelet already contains the second time derivative
	// Compute source wavefield with an additional second-order time derivative
	computeTomoSrcWfldDt2_3D(dev_sourcesSignals[iGpu], wavefield1, dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/************************* Preliminary steps ******************************/
	/**************************************************************************/
	// Copy model to device
	cuda_call(hipMemcpy(dev_modelTomo[iGpu], model, host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Scale model (background perturbation) by 2/v^3 x v^2dtw^2
	scaleReflectivity_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

	// Scale extended reflectivity by 2/v^3 (linearization of wave-equation)
	for (int iExt2=0; iExt2<host_nExt2; iExt2++){
		long long extStride2 = iExt2 * host_extStride;
		for (int iExt1=0; iExt1<host_nExt1; iExt1++){
			long long extStride1 = iExt1 * host_nVel;
			scaleReflectivityLinHxHy_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_extReflectivity[iGpu], dev_reflectivityScale[iGpu], extStride1, extStride2);
		}
	}

	// Allocate and initialize data to zero
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data at coarse time-sampling on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize data on device

	/**************************************************************************/
	/******************************** Leg #1 **********************************/
	/**************************************************************************/
	if (host_leg1 == 1){
		std::cout << "Leg 1 fwd" << std::endl;
		// Source -> reflectivity -> model -> data
		computeTomoLeg1HxHyFwd_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_dataRegDts[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamH2D[iGpu], transferStreamD2H[iGpu]);
	}

	/**************************************************************************/
	/******************************** Leg #2 **********************************/
	/**************************************************************************/
	if (host_leg2 == 1){
		std::cout << "Leg 2 fwd" << std::endl;
		// Source -> model -> reflectivity -> data
		computeTomoLeg2HxHyFwd_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_dataRegDts[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamH2D[iGpu], transferStreamD2H[iGpu]);

	}

	/**************************************************************************/
	/******************************** Data ************************************/
	/**************************************************************************/
	// Copy data to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/**************************** Deallocation ********************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));

}

// Time-lags
void tomoTauShotsFwdGpu_3D(double *model, double *dataRegDts, double *extReflectivity, double *sourcesSignals, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, double *wavefield1, double *wavefield2, int iGpu, int iGpuId){

	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&compStream[iGpu]);
	hipStreamCreate(&transferStreamH2D[iGpu]);
	hipStreamCreate(&transferStreamD2H[iGpu]);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Sources geometry + signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Blocks for Laplacian
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocks data recording
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	/**************************************************************************/
	/****************************** Source ************************************/
	/**************************************************************************/
	// The wavelet already contains the second time derivative
	// Compute source wavefield with an additional second-order time derivative
	computeTomoSrcWfldDt2_3D(dev_sourcesSignals[iGpu], wavefield1, dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/************************* Preliminary steps ******************************/
	/**************************************************************************/
	// Copy model to device
	cuda_call(hipMemcpy(dev_modelTomo[iGpu], model, host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Scale model (background perturbation) by 2/v^3 x v^2dtw^2
	scaleReflectivity_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

	// Scale extended reflectivity by 2/v^3 (linearization of wave-equation)
	for (int iExt2=0; iExt2<host_nExt2; iExt2++){
		long long extStride2 = iExt2 * host_extStride;
		for (int iExt1=0; iExt1<host_nExt1; iExt1++){
			long long extStride1 = iExt1 * host_nVel;
			scaleReflectivityLinHxHy_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_extReflectivity[iGpu], dev_reflectivityScale[iGpu], extStride1, extStride2);
		}
	}

	// Allocate and initialize data to zero
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data at coarse time-sampling on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize data on device

	/**************************************************************************/
	/******************************** Leg #1 **********************************/
	/**************************************************************************/
	if (host_leg1 == 1){
		std::cout << "Leg 1 fwd" << std::endl;
		// Source -> reflectivity -> model -> data
		// computeTomoLeg1TauFwd_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_dataRegDts[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamH2D[iGpu], transferStreamD2H[iGpu]);
	}

	/**************************************************************************/
	/******************************** Leg #2 **********************************/
	/**************************************************************************/
	if (host_leg2 == 1){
		std::cout << "Leg 2 fwd" << std::endl;
		// Source -> model -> reflectivity -> data
		// computeTomoLeg2TauFwd_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_dataRegDts[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamH2D[iGpu], transferStreamD2H[iGpu]);

	}

	/**************************************************************************/
	/******************************** Data ************************************/
	/**************************************************************************/
	// Copy data to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/**************************** Deallocation ********************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));

}

/******************************************************************************/
/************************* Tomo extended adjoint ******************************/
/******************************************************************************/
// Subsurface offsets
void tomoHxHyShotsAdjGpu_3D(double *model, double *dataRegDts, double *extReflectivity, double *sourcesSignals, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, double *wavefield1, double *wavefield2, int iGpu, int iGpuId, double *dataRegDtsQc){

	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&compStream[iGpu]);
	hipStreamCreate(&transferStreamH2D[iGpu]);
	hipStreamCreate(&transferStreamD2H[iGpu]);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Blocks for Laplacian
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocks data recording
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	/**************************************************************************/
	/****************************** Source ************************************/
	/**************************************************************************/

	// The wavelet already contains the second time derivative
	// Compute source wavefield with an additional second-order time derivative
	computeTomoSrcWfldDt2_3D(dev_sourcesSignals[iGpu], wavefield1, dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/****************************** Receiver **********************************/
	/**************************************************************************/
	// Allocate data at coarse time-sampling on device and copy data from host -> device
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double)));
  	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice));

  	cuda_call(hipMalloc((void**) &dev_dataRegDtsQc[iGpu], nReceiversReg*host_nts*sizeof(double)));
	cuda_call(hipMemset(dev_dataRegDtsQc[iGpu], 0, nReceiversReg*host_nts*sizeof(double)));

	// Compute receiver wavefield and store it into wavefield2 on RAM
	computeTomoRecWfld_3D(dev_dataRegDts[iGpu], wavefield2, dev_receiversPositionReg[iGpu], dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/************************* Preliminary steps ******************************/
	/**************************************************************************/
	// Set model to zero
	cuda_call(hipMemset(dev_modelTomo[iGpu], 0, host_nVel*sizeof(double)));

	// Scale extended reflectivity by 2/v^3 (linearization of wave-equation)
	for (int iExt2=0; iExt2<host_nExt2; iExt2++){
		long long extStride2 = iExt2 * host_extStride;
		for (int iExt1=0; iExt1<host_nExt1; iExt1++){
			long long extStride1 = iExt1 * host_nVel;
			scaleReflectivityLinHxHy_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_extReflectivity[iGpu], dev_reflectivityScale[iGpu], extStride1, extStride2);
		}
	}

	/**************************************************************************/
	/******************************** Leg #1 **********************************/
	/**************************************************************************/
	if (host_leg1 == 1){
		std::cout << "Leg 1 adj" << std::endl;
		// Source -> reflectivity -> model -> data
		computeTomoLeg1HxHyAdj_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamH2D[iGpu], nblockData, dev_dataRegDtsQc[iGpu]);
	}

	/**************************************************************************/
	/******************************** Leg #2 **********************************/
	/**************************************************************************/
	if (host_leg2 == 1){
		std::cout << "Leg 2 adj" << std::endl;
		// Source -> reflectivity -> model -> data
		computeTomoLeg2HxHyAdj_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamH2D[iGpu], nblockData, dev_dataRegDtsQc[iGpu]);

	}

	/**************************************************************************/
	/******************************** Model ***********************************/
	/**************************************************************************/
	// Scale model for finite-difference and secondary source coefficient
	scaleReflectivity_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

	// Copy model to host
	cuda_call(hipMemcpy(model, dev_modelTomo[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// QC
	cuda_call(hipMemcpy(dataRegDtsQc, dev_dataRegDtsQc[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/**************************** Deallocation ********************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));

}

// Time-lags
void tomoTauShotsAdjGpu_3D(double *model, double *dataRegDts, double *extReflectivity, double *sourcesSignals, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, double *wavefield1, double *wavefield2, int iGpu, int iGpuId, double *dataRegDtsQc){

	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&compStream[iGpu]);
	hipStreamCreate(&transferStreamH2D[iGpu]);
	hipStreamCreate(&transferStreamD2H[iGpu]);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Blocks for Laplacian
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocks data recording
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	/**************************************************************************/
	/****************************** Source ************************************/
	/**************************************************************************/

	// The wavelet already contains the second time derivative
	// Compute source wavefield with an additional second-order time derivative
	computeTomoSrcWfldDt2_3D(dev_sourcesSignals[iGpu], wavefield1, dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/****************************** Receiver **********************************/
	/**************************************************************************/
	// Allocate data at coarse time-sampling on device and copy data from host -> device
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double)));
  	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice));

  	cuda_call(hipMalloc((void**) &dev_dataRegDtsQc[iGpu], nReceiversReg*host_nts*sizeof(double)));
	cuda_call(hipMemset(dev_dataRegDtsQc[iGpu], 0, nReceiversReg*host_nts*sizeof(double)));

	// Compute receiver wavefield and store it into wavefield2 on RAM
	computeTomoRecWfld_3D(dev_dataRegDts[iGpu], wavefield2, dev_receiversPositionReg[iGpu], dimGrid, dimBlock, nblockData, iGpu, compStream[iGpu], transferStreamD2H[iGpu]);

	/**************************************************************************/
	/************************* Preliminary steps ******************************/
	/**************************************************************************/
	// Set model to zero
	cuda_call(hipMemset(dev_modelTomo[iGpu], 0, host_nVel*sizeof(double)));

	// Scale extended reflectivity by 2/v^3 (linearization of wave-equation)
	for (int iExt2=0; iExt2<host_nExt2; iExt2++){
		long long extStride2 = iExt2 * host_extStride;
		for (int iExt1=0; iExt1<host_nExt1; iExt1++){
			long long extStride1 = iExt1 * host_nVel;
			scaleReflectivityLinHxHy_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_extReflectivity[iGpu], dev_reflectivityScale[iGpu], extStride1, extStride2);
		}
	}

	/**************************************************************************/
	/******************************** Leg #1 **********************************/
	/**************************************************************************/
	if (host_leg1 == 1){
		std::cout << "Leg 1 adj" << std::endl;
		// Source -> reflectivity -> model -> data
		computeTomoLeg1HxHyAdj_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamH2D[iGpu], nblockData, dev_dataRegDtsQc[iGpu]);
	}

	/**************************************************************************/
	/******************************** Leg #2 **********************************/
	/**************************************************************************/
	if (host_leg2 == 1){
		std::cout << "Leg 2 adj" << std::endl;
		// Source -> reflectivity -> model -> data
		computeTomoLeg2HxHyAdj_3D(dev_modelTomo[iGpu], wavefield1, wavefield2, dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, iGpu, compStream[iGpu], transferStreamH2D[iGpu], nblockData, dev_dataRegDtsQc[iGpu]);

	}

	/**************************************************************************/
	/******************************** Model ***********************************/
	/**************************************************************************/
	// Scale model for finite-difference and secondary source coefficient
	scaleReflectivity_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

	// Copy model to host
	cuda_call(hipMemcpy(model, dev_modelTomo[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// QC
	cuda_call(hipMemcpy(dataRegDtsQc, dev_dataRegDtsQc[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/**************************** Deallocation ********************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));

}
/******************************************************************************/
/***************************** Auxiliary functions ****************************/
/******************************************************************************/

/***************************** Common parts ***********************************/
// Source wavefield with an additional second time derivative
void computeTomoSrcWfldDt2_3D(double *dev_sourcesIn, double *wavefield1, long long *dev_sourcesPositionsRegIn, int nSourcesRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamIn){

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize time-slices for transfer to host's pinned memory
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize pinned memory
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));

	double *dummySliceLeft, *dummySliceRight;
	dummySliceLeft = new double[host_nVel];
	dummySliceRight = new double[host_nVel];

	// Compute coarse source wavefield sample at its = 0
	int its = 0;

	// Loop within two values of its (coarse time grid)
	for (int it2 = 1; it2 < host_sub+1; it2++){

		// Compute fine time-step index
		int itw = its * host_sub + it2;

		// Step forward
		stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

		// Inject source
		injectSourceLinear_3D<<<1, nSourcesRegIn, 0, compStreamIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn);

		// Damp wavefields
		dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

		// Spread energy to dev_pLeft and dev_pRight
		interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2);

		// Switch pointers
		dev_temp1[iGpu] = dev_p0[iGpu];
		dev_p0[iGpu] = dev_p1[iGpu];
		dev_p1[iGpu] = dev_temp1[iGpu];
		dev_temp1[iGpu] = NULL;
	}

	// Copy pDt1 (its=0)
	cuda_call(hipMemcpyAsync(dev_pDt1[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Switch pointers
	dev_pTemp[iGpu] = dev_pLeft[iGpu];
	dev_pLeft[iGpu] = dev_pRight[iGpu];
	dev_pRight[iGpu] = dev_pTemp[iGpu];
	dev_pTemp[iGpu] = NULL;
	cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

	/************************** Main loop (its > 0) ***************************/
	for (int its = 1; its < host_nts-1; its++){

		// Loop within two values of its (coarse time grid)
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject source
			injectSourceLinear_3D<<<1, nSourcesRegIn, 0, compStreamIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pLeft and dev_pRight
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Copy source wavefield value at its into pDt2
		cuda_call(hipMemcpyAsync(dev_pDt2[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Compute second-order time-derivative of source wavefield at its-1
	    srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt0[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu]);
		cuda_call(hipStreamSynchronize(compStreamIn));
		cuda_call(hipMemcpy(dummySliceRight, dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

		// Wait for pStream to be free
		cuda_call(hipStreamSynchronize(transferStreamIn));
		cuda_call(hipMemcpyAsync(dev_pStream[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its > 1){
			std::memcpy(wavefield1+(its-2)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));
		}

		//// WHY DO YOU NEED THAT ONE ??? ////
		cuda_call(hipStreamSynchronize(compStreamIn));

		cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamIn));

		// Switch pointers
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for time derivative
		dev_pDtTemp[iGpu] = dev_pDt0[iGpu];
		dev_pDt0[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;

	}

	// Copy source wavefield at nts-1 into pDt2
	cuda_call(hipMemcpyAsync(dev_pDt2[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Compute second order time derivative of source wavefield at nts-2
	srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt0[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu]);

	// Wait until pStream has been transfered to host
	cuda_call(hipStreamSynchronize(transferStreamIn));

	// Copy dev_pSourceWavefield into pStream
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Copy second order time derivative of source wavefield at nts-3 from pin -> RAM
	std::memcpy(wavefield1+(host_nts-3)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

	// Transfer pStream (second order time derivative of source wavefield at nts-2) to pin
	cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamIn));

	// In the meantime, reset pDt0 and compute second order time-derivative at nts-1
	cuda_call(hipMemsetAsync(dev_pDt0[iGpu], 0, host_nVel*sizeof(double), compStreamIn));
	srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu], dev_pDt0[iGpu]);

	// Wait until pStream has been fully transfered to pin (derivative of source wavefield at nts-2)
	cuda_call(hipStreamSynchronize(transferStreamIn));

	// Copy source derivative from pin -> RAM
	std::memcpy(wavefield1+(host_nts-2)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

	// Copy source derivative at nts-1
	cuda_call(hipMemcpy(pin_wavefieldSlice1[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
	std::memcpy(wavefield1+(host_nts-1)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

}

// Receiver wavefield
void computeTomoRecWfld_3D(double *dev_dataRegDtsIn, double *wavefield2, long long *dev_receiversPositionRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamIn){

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize time-slices for transfer to host's pinned memory
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize pinned memory
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));

	// Start propagating adjoint wavefield
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step adjoint
			stepAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject data
			interpLinearInjectData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_dataRegDtsIn, dev_p0[iGpu], its, it2, dev_receiversPositionRegIn);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Interpolate and record time slices of receiver wavefield at coarse sampling (no scaling applied yet)
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Wait until pStream has been transfered
		cuda_call(hipStreamSynchronize(transferStreamIn));

		// Copy pRight (contains wavefield at its+1) into pStream
		cuda_call(hipMemcpyAsync(dev_pStream[iGpu], dev_pRight[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy wavefield at its+1 from pin -> RAM
		if (its < host_nts-2) {
			std::memcpy(wavefield2+(its+2)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

		}

		// Wait until pStream has been updated
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Transfer pStream -> pin (at its+1)
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pLeft[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

	}

	// At this point, pin contains receiver wavefield at its=1

	// Wait until pStream has been transfered
	cuda_call(hipStreamSynchronize(transferStreamIn));

 	// Copy wavefield at its=1 from pin -> RAM
	std::memcpy(wavefield2+host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

	// Transfer pStream -> pin (at its=0)
	cuda_call(hipMemcpy(pin_wavefieldSlice1[iGpu], dev_pRight[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy wavefield at its=0 from pin -> RAM
	std::memcpy(wavefield2, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

}

/******************************** Forward *************************************/
// Source -> reflectivity -> model -> data
void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #1 *************************/
	/*************** Source -> reflectivity -> scattered wavefield ************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
	cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield1 slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Launch transfer of wavefield2 slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Wait until the scattered wavefield has been transfered to pin
		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its>0) {
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));
		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	// Wait until the scattered wavefield has been transfered to pin
	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	// Transfer pDt0 -> pin
	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #1 ************************/
	/***************** Scattered wavefield -> model -> data *******************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// // Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu],wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double));
			cuda_call(hipStreamSynchronize(compStreamIn));
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}
}

// Source -> model -> reflectivity -> data
void computeTomoLeg2HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #2 *************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+1)*host_nVel, host_nVel*sizeof(double));
		cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Asynchronous copy of dev_pDt1 => dev_pDt0 (scattered wavefield at its)
		cuda_call(hipMemcpy(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));
		cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
		std::memcpy(wavefield2+its*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));

	}

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpy(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));
	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #2 ************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);

	// imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);
	// scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);


	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+1)*host_nVel, host_nVel*sizeof(double));
		cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);
		// scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);


		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	}
}

/******************************** Adjoint *************************************/
// Source -> reflectivity -> model <- data
void computeTomoLeg1HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn){

	/************* Compute scattered wavefield and imaging condition **********/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRecWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn));

	// Copy source wavefield slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Copy receiver wavefield slice from RAM -> pinned for time its = 0 -> transfer to pDt0
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy source wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy receiver wavefield value at its from pDt0 -> pRecWavefield
		cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){

			// Copy wavefield1 slice its+2 from RAM -> pin
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));

			// Wait until dev_pStream is ready to be used
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Copy wavefield2 slice its+1 from RAM -> pin
		std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(its+1)*host_nVel, host_nVel*sizeof(double));

		// Wait until dev_pDt0 is ready to be used
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from pin -> dev_pDt0 for receiver wavefield at its+1
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsQcIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// At this point, pDt1 contains the value of the scattered wavefield at its
		// The imaging condition can be done for its

		// Apply imaging condition at its
		imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt1[iGpu], dev_pRecWavefield[iGpu]);

		// Wait until transfer stream has finished copying slice its from pinned -> pStream
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));

	}

	// Copy receiver wavefield value at nts-1 from pDt0 -> pRecWavefield
	cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Compute imaging condition at its = nts-1
	imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt1[iGpu], dev_pRecWavefield[iGpu]);

	// Scale model for finite-difference and secondary source coefficient
	// scaleReflectivity_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

}

// Source -> reflectivity -> model <- data
void computeTomoLeg2HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn){

	/************* Compute scattered wavefield and imaging condition **********/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRecWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Copy receiver wavefield time-slice its = nts-1
	// From RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(host_nts-1)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pRecWavefield[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRecWavefield[iGpu], dev_vel2Dtw2[iGpu]);

	// Compute secondary source for its = nts-1
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyTomoAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_pRecWavefield[iGpu], dev_extReflectivityIn, ihx, iExt1, ihy, iExt2);
		}
	}

	// Copy receiver wavefield slice from RAM -> pinned for time nts-2 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(host_nts-2)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Copy source wavefield slice from RAM -> pinned for time its = nts-1 -> transfer to pDt0
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(host_nts-1)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = host_nts-2; its > -1; its--){

		// Copy receiver wavefield value at its from pDt0 -> pRecWavefield
		cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy receiver wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its > 0){

			// Copy receiver wavefield slice its-1 from RAM -> pin
			std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(its-1)*host_nVel, host_nVel*sizeof(double));

			// Wait until dev_pStream is ready to be used
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load receiver wavefield slice its-1 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Copy source wavefield slice its from RAM -> pin
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+its*host_nVel, host_nVel*sizeof(double));

		// Wait until dev_pDt0 is ready to be used
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from pin -> dev_pDt0 for receiver wavefield at its
		cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRecWavefield[iGpu], dev_vel2Dtw2[iGpu]);

		// Compute secondary source for its = nts-1
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyTomoAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRecWavefield[iGpu], dev_extReflectivityIn, ihx, iExt1, ihy, iExt2);
			}
		}

		// Start subloop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step forward
			stepAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2+1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Apply imaging condition at its+1
		imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt2[iGpu], dev_pSourceWavefield[iGpu]);

		// Wait until transfer stream has finished copying slice its from pinned -> pStream
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));

	}

	// Copy receiver wavefield value at its = 0 from pStream -> pSourceWavefield
	cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Compute imaging condition at its = 0
	imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt2[iGpu], dev_pSourceWavefield[iGpu]);

}
