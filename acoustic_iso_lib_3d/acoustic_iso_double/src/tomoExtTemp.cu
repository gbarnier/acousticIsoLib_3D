/******************************** Leg 1 ***************************************/
// Source -> reflectivity -> model -> data
// void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){
//
// 	/**************************************************************************/
// 	/*************************** First part of leg #1 *************************/
// 	/**************************************************************************/
//
// 	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamIn));
			// Wait until pStream is ready to be updated
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its>0) {
			// Standard library
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn)); // Reinitialize dev_pRight to zero (because of the += in the kernel)

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// In the meantime, copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));
//
// 	/**************************************************************************/
// 	/*************************** First part of leg #1 *************************/
// 	/**************************************************************************/
//
// 	// Scatter wavefield2 on model perturbation
//
// 	// Reset the time slices to zero
// 	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
//
// 	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
// 	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
// 	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
//
// 	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
// 	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);
//
// 	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
// 	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
// 	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
// 	cuda_call(hipStreamSynchronize(transferStreamH2DIn));
//
// 	// At that point:
// 	// dev_pSourceWavefield contains wavefield at its=1
// 	// pin_wavefieldSlice and dev_pStream are free to be used
// 	// dev_pLeft (secondary source at its = 0) is computed
//
// 	// Start propagating scattered wavefield
// 	for (int its = 0; its < host_nts-1; its++){
//
// 		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
// 		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));
//
// 		if (its < host_nts-2){
// 			// Copy wavefield slice its+2 from RAM > dev_pStream
// 			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double)); // -> this should be done with transfer stream
// 			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamH2DIn));
// 			cuda_call(hipStreamSynchronize(compStreamIn));
// 			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
// 		}
//
// 		// Compute secondary source for first coarse time index (its+1) with compute stream
// 		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);
//
// 		for (int it2 = 1; it2 < host_sub+1; it2++){
//
// 			// Step forward
// 			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
//
// 			// Inject secondary source sample itw-1
// 			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);
//
// 			// Damp wavefields
// 			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);
//
// 			// Extract data
// 			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);
//
// 			// Switch pointers
// 			dev_temp1[iGpu] = dev_p0[iGpu];
// 			dev_p0[iGpu] = dev_p1[iGpu];
// 			dev_p1[iGpu] = dev_temp1[iGpu];
// 			dev_temp1[iGpu] = NULL;
//
// 		}
//
// 		// Switch pointers for secondary source
// 		dev_pTemp[iGpu] = dev_pLeft[iGpu];
// 		dev_pLeft[iGpu] = dev_pRight[iGpu];
// 		dev_pRight[iGpu] = dev_pTemp[iGpu];
// 		dev_pTemp[iGpu] = NULL;
// 		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nModel*sizeof(double), compStreamIn));
//
// 		// Wait until the transfer from pinned -> pStream is completed
// 		cuda_call(hipStreamSynchronize(transferStreamH2DIn));
// 	}
//
// }

void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #1 *************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamIn));
			// Wait until pStream is ready to be updated
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its>0) {
			// Standard library
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn)); // Reinitialize dev_pRight to zero (because of the += in the kernel)

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// In the meantime, copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #1 ************************/
	/**************************************************************************/

	// Scatter wavefield2 on model perturbation

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
	cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	// At that point:
	// dev_pSourceWavefield contains wavefield at its=1
	// pin_wavefieldSlice and dev_pStream are free to be used
	// dev_pLeft (secondary source at its = 0) is computed

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double)); // -> this should be done with transfer stream
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamH2DIn));
			cuda_call(hipStreamSynchronize(compStreamIn));
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nModel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));
	}

}

// Source -> reflectivity -> model <- data
void computeTomoLeg1HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn){

	/************* Compute scattered wavefield and imaging condition **********/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRecWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy source wavefield slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Copy receiver wavefield slice from RAM -> pinned for time its = 0 -> transfer to pDt0
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy source wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy receiver wavefield value at its from pDt0 -> pRecWavefield
		cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){

			// Copy wavefield1 slice its+2 from RAM -> pin
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));

			// Wait until dev_pStream is ready to be used
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Copy wavefield2 slice its+1 from RAM -> pin
		std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(its+1)*host_nVel, host_nVel*sizeof(double));

		// Wait until dev_pDt0 is ready to be used
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from pin -> dev_pDt0 for receiver wavefield at its+1
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsQcIn, its, it2, dev_receiversPositionRegIn);

			// Spread energy to dev_pLeft and dev_pRight
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_t



////////////////////////////////////////////////////////////////////////////////
// Leg 2 tomo forward
// Source -> model -> reflectivity -> data
void computeTomoLeg2HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #2 *************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
	cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield1 slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Launch transfer of wavefield2 slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Wait until the scattered wavefield has been transfered to pin
		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its > 0) {
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));
		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	// Wait until the scattered wavefield has been transfered to pin
	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	// Transfer pDt0 -> pin
	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #2 ************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu],wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double));
			cuda_call(hipStreamSynchronize(compStreamIn));
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);


		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}
}

// Leg 2 tomo adjoint
// Source -> reflectivity -> model <- data
void computeTomoLeg2HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn){

	/************* Compute scattered wavefield and imaging condition **********/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRecWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Copy receiver wavefield time-slice its = nts-1
	// From RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(host_nts-1)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pRecWavefield[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRecWavefield[iGpu], dev_vel2Dtw2[iGpu]);

	// Compute secondary source for its = nts-1
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyTomoAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_pRecWavefield[iGpu], dev_extReflectivityIn, ihx, iExt1, ihy, iExt2);
		}
	}

	// Copy receiver wavefield slice from RAM -> pinned for time nts-2 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(host_nts-2)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Copy source wavefield slice from RAM -> pinned for time its = nts-1 -> transfer to pDt0
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(host_nts-1)*host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = host_nts-2; its > -1; its--){

		// Copy receiver wavefield value at its from pDt0 -> pRecWavefield
		cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy receiver wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its > 0){

			// Copy receiver wavefield slice its-1 from RAM -> pin
			std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(its-1)*host_nVel, host_nVel*sizeof(double));

			// Wait until dev_pStream is ready to be used
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load receiver wavefield slice its-1 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Copy source wavefield slice its from RAM -> pin
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+its*host_nVel, host_nVel*sizeof(double));

		// Wait until dev_pDt0 is ready to be used
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from pin -> dev_pDt0 for receiver wavefield at its
		cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRecWavefield[iGpu], dev_vel2Dtw2[iGpu]);
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Compute secondary source for its = nts-1
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyTomoAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRecWavefield[iGpu], dev_extReflectivityIn, ihx, iExt1, ihy, iExt2);
			}
		}

		// Start subloop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step forward
			stepAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2+1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// At this point, pDt1 contains the value of the scattered wavefield at its
		// The imaging condition can be done for its

		// Apply imaging condition at its+1
		imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt2[iGpu], dev_pSourceWavefield[iGpu]);

		// Wait until transfer stream has finished copying slice its from pinned -> pStream
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));

	}

	// Copy receiver wavefield value at its = 0 from pStream -> pSourceWavefield
	cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Compute imaging condition at its = nts-1
	imagingAdjGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pDt2[iGpu], dev_pSourceWavefield[iGpu]);

	// Scale model for finite-difference and secondary source coefficient
	// scaleReflectivity_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]);

}


void computeTomoSrcWfldDt2_3D(double *dev_sourcesIn, double *wavefield1, long long *dev_sourcesPositionsRegIn, int nSourcesRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamIn){

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize time-slices for transfer to host's pinned memory
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));

	// Initialize pinned memory
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));

	double *dummySliceLeft, *dummySliceRight;
	dummySliceLeft = new double[host_nVel];
	dummySliceRight = new double[host_nVel];

	// Compute coarse source wavefield sample at its = 0
	int its = 0;

	// Loop within two values of its (coarse time grid)
	for (int it2 = 1; it2 < host_sub+1; it2++){

		// Compute fine time-step index
		int itw = its * host_sub + it2;

		// Step forward
		stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

		// Inject source
		injectSourceLinear_3D<<<1, nSourcesRegIn, 0, compStreamIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn);

		// Damp wavefields
		dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

		// Spread energy to dev_pLeft and dev_pRight
		interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2);

		// Switch pointers
		dev_temp1[iGpu] = dev_p0[iGpu];
		dev_p0[iGpu] = dev_p1[iGpu];
		dev_p1[iGpu] = dev_temp1[iGpu];
		dev_temp1[iGpu] = NULL;
	}

	// Copy pDt1 (its=0)
	cuda_call(hipMemcpyAsync(dev_pDt1[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Switch pointers
	dev_pTemp[iGpu] = dev_pLeft[iGpu];
	dev_pLeft[iGpu] = dev_pRight[iGpu];
	dev_pRight[iGpu] = dev_pTemp[iGpu];
	dev_pTemp[iGpu] = NULL;
	cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

	/************************** Main loop (its > 0) ***************************/
	for (int its = 1; its < host_nts-1; its++){

		// Loop within two values of its (coarse time grid)
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject source
			injectSourceLinear_3D<<<1, nSourcesRegIn, 0, compStreamIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pLeft and dev_pRight
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Copy source wavefield value at its into pDt2
		cuda_call(hipMemcpyAsync(dev_pDt2[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Compute second-order time-derivative of source wavefield at its-1
	    srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt0[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu]);
		cuda_call(hipStreamSynchronize(compStreamIn));
		cuda_call(hipMemcpy(dummySliceRight, dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

		// Wait for pStream to be free
		cuda_call(hipStreamSynchronize(transferStreamIn));
		cuda_call(hipMemcpyAsync(dev_pStream[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its > 1){
			std::memcpy(wavefield1+(its-2)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));
		}

		//// WHY DO YOU NEED THAT ONE ??? ////
		cuda_call(hipStreamSynchronize(compStreamIn));

		cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamIn));

		// Switch pointers
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for time derivative
		dev_pDtTemp[iGpu] = dev_pDt0[iGpu];
		dev_pDt0[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;

	}

	// Copy source wavefield at nts-1 into pDt2
	cuda_call(hipMemcpyAsync(dev_pDt2[iGpu], dev_pLeft[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Compute second order time derivative of source wavefield at nts-2
	srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt0[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu]);

	// Wait until pStream has been transfered to host
	cuda_call(hipStreamSynchronize(transferStreamIn));

	// Copy dev_pSourceWavefield into pStream
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// Copy second order time derivative of source wavefield at nts-3 from pin -> RAM
	std::memcpy(wavefield1+(host_nts-3)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

	// Transfer pStream (second order time derivative of source wavefield at nts-2) to pin
	cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamIn));

	// In the meantime, reset pDt0 and compute second order time-derivative at nts-1
	cuda_call(hipMemsetAsync(dev_pDt0[iGpu], 0, host_nVel*sizeof(double), compStreamIn));
	srcWfldSecondTimeDerivative_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pSourceWavefield[iGpu], dev_pDt1[iGpu], dev_pDt2[iGpu], dev_pDt0[iGpu]);

	// Wait until pStream has been fully transfered to pin (derivative of source wavefield at nts-2)
	cuda_call(hipStreamSynchronize(transferStreamIn));

	// Copy source derivative from pin -> RAM
	std::memcpy(wavefield1+(host_nts-2)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

	// Copy source derivative at nts-1
	cuda_call(hipMemcpy(pin_wavefieldSlice1[iGpu], dev_pSourceWavefield[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
	std::memcpy(wavefield1+(host_nts-1)*host_nVel, pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double));

}


// Source -> model -> reflectivity -> data
void computeTomoLeg2HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #2 *************************/
	/******************** Source -> model -> scattered wavefield **************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+1)*host_nVel, host_nVel*sizeof(double));
		cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Asynchronous copy of dev_pDt1 => dev_pDt0 (scattered wavefield at its)
		cuda_call(hipMemcpy(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));
		cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
		std::memcpy(wavefield2+its*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));

	}

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpy(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice));
	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #2 ************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);

	// imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);
	// scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);


	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+1)*host_nVel, host_nVel*sizeof(double));
		cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);
		// scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);


		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	}
}
