/******************************** Leg 1 ***************************************/
// Source -> reflectivity -> model -> data
// void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){
//
// 	/**************************************************************************/
// 	/*************************** First part of leg #1 *************************/
// 	/**************************************************************************/
//
// 	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamIn));
			// Wait until pStream is ready to be updated
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its>0) {
			// Standard library
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn)); // Reinitialize dev_pRight to zero (because of the += in the kernel)

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// In the meantime, copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));
//
// 	/**************************************************************************/
// 	/*************************** First part of leg #1 *************************/
// 	/**************************************************************************/
//
// 	// Scatter wavefield2 on model perturbation
//
// 	// Reset the time slices to zero
// 	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
// 	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
//
// 	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
// 	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
// 	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));
//
// 	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
// 	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);
//
// 	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
// 	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
// 	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
// 	cuda_call(hipStreamSynchronize(transferStreamH2DIn));
//
// 	// At that point:
// 	// dev_pSourceWavefield contains wavefield at its=1
// 	// pin_wavefieldSlice and dev_pStream are free to be used
// 	// dev_pLeft (secondary source at its = 0) is computed
//
// 	// Start propagating scattered wavefield
// 	for (int its = 0; its < host_nts-1; its++){
//
// 		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
// 		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));
//
// 		if (its < host_nts-2){
// 			// Copy wavefield slice its+2 from RAM > dev_pStream
// 			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double)); // -> this should be done with transfer stream
// 			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamH2DIn));
// 			cuda_call(hipStreamSynchronize(compStreamIn));
// 			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
// 		}
//
// 		// Compute secondary source for first coarse time index (its+1) with compute stream
// 		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);
//
// 		for (int it2 = 1; it2 < host_sub+1; it2++){
//
// 			// Step forward
// 			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
//
// 			// Inject secondary source sample itw-1
// 			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);
//
// 			// Damp wavefields
// 			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);
//
// 			// Extract data
// 			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);
//
// 			// Switch pointers
// 			dev_temp1[iGpu] = dev_p0[iGpu];
// 			dev_p0[iGpu] = dev_p1[iGpu];
// 			dev_p1[iGpu] = dev_temp1[iGpu];
// 			dev_temp1[iGpu] = NULL;
//
// 		}
//
// 		// Switch pointers for secondary source
// 		dev_pTemp[iGpu] = dev_pLeft[iGpu];
// 		dev_pLeft[iGpu] = dev_pRight[iGpu];
// 		dev_pRight[iGpu] = dev_pTemp[iGpu];
// 		dev_pTemp[iGpu] = NULL;
// 		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nModel*sizeof(double), compStreamIn));
//
// 		// Wait until the transfer from pinned -> pStream is completed
// 		cuda_call(hipStreamSynchronize(transferStreamH2DIn));
// 	}
//
// }

void computeTomoLeg1HxHyFwd_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_dataRegDtsIn, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, hipStream_t transferStreamD2HIn){

	/**************************************************************************/
	/*************************** First part of leg #1 *************************/
	/**************************************************************************/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double));
	hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamIn));
			// Wait until pStream is ready to be updated
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Spread energy to dev_pDt1 and dev_pDt2
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Extract data
			// recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsIn, its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		cuda_call(hipStreamSynchronize(transferStreamD2HIn));

		// Asynchronous copy of dev_pDt1 => dev_pDt0 [its] [compute]
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Save wavefield2 from pin -> RAM for its-1
		if (its>0) {
			// Standard library
			std::memcpy(wavefield2+(its-1)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

		}

		// Wait until pDt0 is ready to be transfered
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from device -> host of wavefield2 at its
		cuda_call(hipMemcpyAsync(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost, transferStreamD2HIn));

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nVel*sizeof(double), compStreamIn));

		// Switch pointers for the scattered wavefield
		dev_pDtTemp[iGpu] = dev_pDt1[iGpu];
		dev_pDt1[iGpu] = dev_pDt2[iGpu];
		dev_pDt2[iGpu] = dev_pDtTemp[iGpu];
		dev_pDtTemp[iGpu] = NULL;
  		cuda_call(hipMemsetAsync(dev_pDt2[iGpu], 0, host_nVel*sizeof(double), compStreamIn)); // Reinitialize dev_pRight to zero (because of the += in the kernel)

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	}

	cuda_call(hipStreamSynchronize(transferStreamD2HIn));

	// Load pLeft to pStream (value of wavefield at nts-1)
	cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], dev_pDt1[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

	// In the meantime, copy value of wavefield at nts-2 from pinned memory to RAM
	std::memcpy(wavefield2+(host_nts-2)*host_nVel, pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	// Wait until pDt1 -> pDt0 is done
	cuda_call(hipStreamSynchronize(compStreamIn));

	cuda_call(hipMemcpy(pin_wavefieldSlice2[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToHost));

	// Copy pinned -> RAM
	std::memcpy(wavefield2+(host_nts-1)*host_nVel,pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double));

	/**************************************************************************/
	/*************************** Second part of leg #1 ************************/
	/**************************************************************************/

	// Scatter wavefield2 on model perturbation

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRight[iGpu], 0, host_nVel*sizeof(double)));

	// Copy wavefield2 time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu]);

	// Copy new slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
	cuda_call(hipStreamSynchronize(transferStreamH2DIn));

	// At that point:
	// dev_pSourceWavefield contains wavefield at its=1
	// pin_wavefieldSlice and dev_pStream are free to be used
	// dev_pLeft (secondary source at its = 0) is computed

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){
			// Copy wavefield slice its+2 from RAM > dev_pStream
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double)); // -> this should be done with transfer stream
			// cuda_call(hipMemcpyAsync(pin_wavefieldSlice[iGpu], wavefield2+(its+2)*host_nVel, host_nVel*sizeof(double), hipMemcpyHostToHost, transferStreamH2DIn));
			cuda_call(hipStreamSynchronize(compStreamIn));
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Compute secondary source for first coarse time index (its+1) with compute stream
		imagingFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_modelTomoIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu]);

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionRegIn);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_pTemp[iGpu] = dev_pLeft[iGpu];
		dev_pLeft[iGpu] = dev_pRight[iGpu];
		dev_pRight[iGpu] = dev_pTemp[iGpu];
		dev_pTemp[iGpu] = NULL;
		cuda_call(hipMemsetAsync(dev_pRight[iGpu], 0, host_nModel*sizeof(double), compStreamIn));

		// Wait until the transfer from pinned -> pStream is completed
		cuda_call(hipStreamSynchronize(transferStreamH2DIn));
	}

}

// Source -> reflectivity -> model <- data
void computeTomoLeg1HxHyAdj_3D(double *dev_modelTomoIn, double *wavefield1, double *wavefield2, double *dev_extReflectivityIn, long long *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, hipStream_t compStreamIn, hipStream_t transferStreamH2DIn, int nBlockDataIn, double *dev_dataRegDtsQcIn){

	/************* Compute scattered wavefield and imaging condition **********/

	// Reset the time slices to zero
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pLeft[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt0[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pDt2[iGpu], 0, host_nVel*sizeof(double)));
  	cuda_call(hipMemset(dev_pStream[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice1[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(pin_wavefieldSlice2[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pSourceWavefield[iGpu], 0, host_nVel*sizeof(double)));
	cuda_call(hipMemset(dev_pRecWavefield[iGpu], 0, host_nVel*sizeof(double)));

	// Copy source wavefield time-slice its = 0: RAM -> pinned -> dev_pSourceWavefield
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pSourceWavefield[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Do first fwd imaging condition for its = 0 (after that, secondary source at its = 0 is done)
	for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
		long long iExt2 = ihy + host_hExt2;
		for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
			long long iExt1 = ihx + host_hExt1;
			imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pLeft[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
		}
	}

	// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
	scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_vel2Dtw2[iGpu]);
	cuda_call(hipStreamSynchronize(compStreamIn)); // ?

	// Copy source wavefield slice from RAM -> pinned for time its = 1 -> transfer to pStream
	std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+host_nVel, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Copy receiver wavefield slice from RAM -> pinned for time its = 0 -> transfer to pDt0
	std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2, host_nVel*sizeof(double));
	cuda_call(hipMemcpy(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice));

	// Start propagating scattered first scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Copy source wavefield value at its+1 from pStream -> pSourceWavefield
		cuda_call(hipMemcpyAsync(dev_pSourceWavefield[iGpu], dev_pStream[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		// Copy receiver wavefield value at its from pDt0 -> pRecWavefield
		cuda_call(hipMemcpyAsync(dev_pRecWavefield[iGpu], dev_pDt0[iGpu], host_nVel*sizeof(double), hipMemcpyDeviceToDevice, compStreamIn));

		if (its < host_nts-2){

			// Copy wavefield1 slice its+2 from RAM -> pin
			std::memcpy(pin_wavefieldSlice1[iGpu], wavefield1+(its+2)*host_nVel, host_nVel*sizeof(double));

			// Wait until dev_pStream is ready to be used
			cuda_call(hipStreamSynchronize(compStreamIn));

			// Load wavefield slice its+2 by transfering from host to device
			cuda_call(hipMemcpyAsync(dev_pStream[iGpu], pin_wavefieldSlice1[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));
		}

		// Copy wavefield2 slice its+1 from RAM -> pin
		std::memcpy(pin_wavefieldSlice2[iGpu], wavefield2+(its+1)*host_nVel, host_nVel*sizeof(double));

		// Wait until dev_pDt0 is ready to be used
		cuda_call(hipStreamSynchronize(compStreamIn));

		// Launch transfer from pin -> dev_pDt0 for receiver wavefield at its+1
		cuda_call(hipMemcpyAsync(dev_pDt0[iGpu], pin_wavefieldSlice2[iGpu], host_nVel*sizeof(double), hipMemcpyHostToDevice, transferStreamH2DIn));

		// Compute secondary source for first coarse time index (its+1) with compute stream
		for (int ihy = -host_hExt2; ihy <= host_hExt2; ihy++){
			long long iExt2 = ihy + host_hExt2;
			for (int ihx = -host_hExt1; ihx <= host_hExt1; ihx++){
				long long iExt1 = ihx + host_hExt1;
				imagingHxHyFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_extReflectivityIn, dev_pRight[iGpu], dev_pSourceWavefield[iGpu], ihx, iExt1, ihy, iExt2);
			}
		}

		// Apply second scaling to secondary source: v^2 * dtw^2 coming from the finite difference scheme
		scaleSecondarySourceFd_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pRight[iGpu], dev_vel2Dtw2[iGpu]);

		// Start subloop
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			stepFwdGpu_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);

			// Inject secondary source sample itw-1
			injectSecondarySource_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pLeft[iGpu], dev_pRight[iGpu], dev_p0[iGpu], it2-1);

			// Damp wavefields
			dampCosineEdge_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_p0[iGpu], dev_p1[iGpu]);

			// Extract data
			recordLinearInterpData_3D<<<nBlockDataIn, BLOCK_SIZE_DATA, 0, compStreamIn>>>(dev_p0[iGpu], dev_dataRegDtsQcIn, its, it2, dev_receiversPositionRegIn);

			// Spread energy to dev_pLeft and dev_pRight
			interpFineToCoarseSlice_3D<<<dimGridIn, dimBlockIn, 0, compStreamIn>>>(dev_pDt1[iGpu], dev_pDt2[iGpu], dev_p0[iGpu], it2);

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_t
