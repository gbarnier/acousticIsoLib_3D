#include "hip/hip_runtime.h"
#include "varDeclare_3D.h"
#include <stdio.h>

/******************************************************************************/
/******************************* Debug Laplacian ******************************/
/******************************************************************************/
//
__global__ void LaplacianFwdGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_model[iGlobalTemp]; // iy = 0
    dev_c_y[2] = dev_model[iGlobalTemp+=yStride]; // iy = 1
    dev_c_y[3] = dev_model[iGlobalTemp+=yStride]; // iy = 2
    shared_c[ixLocal][izLocal] = dev_model[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
    dev_c_y[5] = dev_model[iGlobalTemp+=yStride]; // iy = 4
    dev_c_y[6] = dev_model[iGlobalTemp+=yStride]; // iy = 5
    dev_c_y[7] = dev_model[iGlobalTemp+=yStride];// iy = 6
    dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5]; // Store the middle one in the shared memory (it will be re-used to compute the Laplacian in the z- and x-directions)
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // The last point of the stencil now points to the next y-slice

        // Remark on assignments just above:
        // iyTemp = iy + FAT
        // This guy points to the iy with the largest y-index needed to compute the Laplacian at the new y-position

        // Load the halos in the x-direction
        // Threads with x-index ranging from 0,...,FAT will load the first and last FAT elements of the block on the x-axis to shared memory
        if (threadIdx.y < FAT) {
			shared_c[threadIdx.y][izLocal] = dev_model[iGlobal-dev_nz*FAT]; // Left side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_model[iGlobal+dev_nz*BLOCK_SIZE_X]; // Right side
    	}

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
    		shared_c[ixLocal][threadIdx.x] = dev_model[iGlobal-FAT]; // Up
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_model[iGlobal+BLOCK_SIZE_Z]; // Down
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads(); // Synchronise all threads within each block
    	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
    	// We can now compute the Laplacian value at each point of the entire block slice

        // Apply forward stepping operator
        dev_data[iGlobal] = dev_vel2Dtw2[iGlobal] * (

            dev_coeff[C0] * shared_c[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

/* Forward stepper (no damping) */
__global__ void LaplacianAdjGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];
	double dev_vel_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_data[iGlobalTemp]; // iy = 0
	dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[2] = dev_data[iGlobalTemp+=yStride]; // iy = 1
	dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[3] = dev_data[iGlobalTemp+=yStride]; // iy = 2
	dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];

    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
	shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[5] = dev_data[iGlobalTemp+=yStride]; // iy = 4
	dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[6] = dev_data[iGlobalTemp+=yStride]; // iy = 5
	dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[7] = dev_data[iGlobalTemp+=yStride];// iy = 6
	dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[8] = dev_data[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7
	dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];
		dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];
		dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];
		dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5];
		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];
		dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];
		dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];
		dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];
		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];
    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

        // Apply adjoint stepping operator
        dev_model[iGlobal] = (

            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])
        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;
    }
}

/******************************************************************************/
/******************************** Imaging kernels *****************************/
/******************************************************************************/
// Forward non-extended
__global__ void imagingFwdGpu_3D_zLoop(double *dev_model, double *dev_data, double *dev_sourceWavefieldDts) {

	int iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Coordinate on y-axis

	// Make sure you are inside FAT and ny-FAT (non-included)
	if (iyGlobal < dev_ny-FAT){

		int ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Coordinate on x-axis
		int iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz; // Global coordinate on the time slice
		// Loop over z-axis
		for (int iz=FAT; iz<dev_nz-FAT; iz++){
			dev_data[iGlobal] = dev_model[iGlobal] * dev_sourceWavefieldDts[iGlobal];
			iGlobal=iGlobal+1;
		}
	}
}

__global__ void imagingFwdGpu_3D_yLoop(double *dev_model, double *dev_data, double *dev_sourceWavefieldDts) {

	// Global coordinates for the faster two axes (z and x)
	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate
    int yStride = dev_nz * dev_nx;
    int iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal; // Global position on the cube

    for (int iy=FAT; iy<dev_ny-FAT; iy++){
		dev_data[iGlobal] = dev_model[iGlobal] * dev_sourceWavefieldDts[iGlobal];
		iGlobal+=yStride;
	}
}



	// sum1 = 0.0;
	// sum2 = 0.0;
	// sum3 = 0.0;
	//
	// for (int iy=0; iy<_fdParam_3D->_ny; iy++){
	// 	for (int ix=0; ix<_fdParam_3D->_nx; ix++){
	// 		for (int iz=0; iz<5; iz++){
	// 			sum1 += (*p0->_mat)[iy][ix][iz];
	// 			sum2 += (*p1->_mat)[iy][ix][iz];
	// 			sum3 += (*p2->_mat)[iy][ix][iz];
	// 		}
	// 	}
	// }
	// std::cout << "sum1 forward: " << sum1 << std::endl;
	// std::cout << "sum2 forward: " << sum2 << std::endl;
	// std::cout << "sum3 forward: " << sum3 << std::endl;

	// int index;
	// for (int iy=0; iy<_fdParam_3D->_ny;iy++){
	// 	for (int ix=0; ix<_fdParam_3D->_nx;ix++){
	// 		index = iy * _fdParam_3D->_nx * _fdParam_3D->_nz + ix * _fdParam_3D->_nz + 4;
	// 		std::cout << "iy = " << iy << std::endl;
	// 		std::cout << "ix = " << ix << std::endl;
	// 		for (int its=0; its<_fdParam_3D->_nts; its++){
	// 			std::cout << "nts = " << its << std::endl;
	// 			std::cout << "Data at free surface = " << (*dataRegDts->_mat)[0][its] << std::endl;
	// 		}
	// 	}
	// }

// _freeSurfaceDebugOpObj = std::make_shared<freeSurfaceDebugOp>(_fdParam_3D->_vel, _fdParam_3D->_par, _nGpu, _iGpu, _iGpuId, _iGpuAlloc);

	// std::shared_ptr<double3DReg> modelTest(new double3DReg(_fdParam_3D->_nz, _fdParam_3D->_nx, _fdParam_3D->_ny));
	// std::shared_ptr<double3DReg> dataTest(new double3DReg(_fdParam_3D->_nz, _fdParam_3D->_nx, _fdParam_3D->_ny));
	//
	// _freeSurfaceDebugOpObj(_fdParam_3D->_vel, _fdParam_3D->_par, _nGpu, _iGpu, _iGpuId, _iGpuAlloc);

	// _freeSurfaceDebugOpObj->dotTest(true);
	// _freeSurfaceDebugOpObj->dotTest(true);
	// _freeSurfaceDebugOpObj->dotTest(true);
	// exit(0);

/******************************************************************************/
/******************************* Debug Laplacian ******************************/
/******************************************************************************/
__global__ void LaplacianFwdGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_model[iGlobalTemp]; // iy = 0
    dev_c_y[2] = dev_model[iGlobalTemp+=yStride]; // iy = 1
    dev_c_y[3] = dev_model[iGlobalTemp+=yStride]; // iy = 2
    shared_c[ixLocal][izLocal] = dev_model[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
    dev_c_y[5] = dev_model[iGlobalTemp+=yStride]; // iy = 4
    dev_c_y[6] = dev_model[iGlobalTemp+=yStride]; // iy = 5
    dev_c_y[7] = dev_model[iGlobalTemp+=yStride];// iy = 6
    dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5]; // Store the middle one in the shared memory (it will be re-used to compute the Laplacian in the z- and x-directions)
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // The last point of the stencil now points to the next y-slice

        // Remark on assignments just above:
        // iyTemp = iy + FAT
        // This guy points to the iy with the largest y-index needed to compute the Laplacian at the new y-position

        // Load the halos in the x-direction
        // Threads with x-index ranging from 0,...,FAT will load the first and last FAT elements of the block on the x-axis to shared memory
        if (threadIdx.y < FAT) {
			shared_c[threadIdx.y][izLocal] = dev_model[iGlobal-dev_nz*FAT]; // Left side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_model[iGlobal+dev_nz*BLOCK_SIZE_X]; // Right side
    	}

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
    		shared_c[ixLocal][threadIdx.x] = dev_model[iGlobal-FAT]; // Up
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_model[iGlobal+BLOCK_SIZE_Z]; // Down
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads(); // Synchronise all threads within each block
    	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
    	// We can now compute the Laplacian value at each point of the entire block slice

        // Apply forward stepping operator
        dev_data[iGlobal] = dev_vel2Dtw2[iGlobal] * (

            dev_coeff[C0] * shared_c[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

__global__ void LaplacianAdjGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];
	double dev_vel_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_data[iGlobalTemp]; // iy = 0
	dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[2] = dev_data[iGlobalTemp+=yStride]; // iy = 1
	dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[3] = dev_data[iGlobalTemp+=yStride]; // iy = 2
	dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];

    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
	shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[5] = dev_data[iGlobalTemp+=yStride]; // iy = 4
	dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[6] = dev_data[iGlobalTemp+=yStride]; // iy = 5
	dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[7] = dev_data[iGlobalTemp+=yStride];// iy = 6
	dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[8] = dev_data[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7
	dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];
		dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];
		dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];
		dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5];
		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];
		dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];
		dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];
		dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];
		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];
    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

        // Apply adjoint stepping operator
        dev_model[iGlobal] = (

            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])
        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;
    }
}

__global__ void LaplacianAdjTopGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1]; // Array for the current wavefield y-slice
    double dev_vel_y[2*FAT+1]; // Array for the scaled velocity y-slice

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction (Remember: each thread has its own version of dev_c_y and dev_vel_y array)
    // Points from the current wavefield time-slice that will be used by the current block
    dev_c_y[1] = dev_data[iGlobalTemp];								dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[2] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[3] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];
	// These ones go to shared memory because used multiple times in Laplacian computation for the z- and x-directions
    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride]; 		shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];
	dev_c_y[5] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[6] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];
	dev_c_y[7] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[8] = dev_data[iGlobalTemp+=yStride];						dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iyGlobal=FAT; iyGlobal<dev_ny-FAT; iyGlobal++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];						dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];						dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];						dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block before updating the value of the shared memory at ixLocal, izLocal
        shared_c[ixLocal][izLocal] = dev_c_y[5]; 		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];						dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];						dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];						dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];
    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

        // Apply adjoint stepping operator
		if (izGlobal <= 4){
			dev_model[iGlobal] = 0.0;
		}
		if (izGlobal == 5){
	        dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
	            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ2] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]
				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
	            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

	        );

	        // Move forward one grid point in the y-direction
	        iGlobal = iGlobal + yStride;
		}

		if (izGlobal == 6){

			dev_model[iGlobal] = (

				dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

				+ dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
				+ dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
				+ dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

				+ dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
				+ dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
				+ dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

				+ dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
				+ dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
				+ dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

				+ dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
				+ dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
				+ dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

			);

		}
		if (izGlobal == 7){

			dev_model[iGlobal] = (

				dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

				+ dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
				+ dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
				+ dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2]

				+ dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
				+ dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
				+ dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

				+ dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
				+ dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
				+ dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

				+ dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
				+ dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
				+ dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

			);

		}
		if (izGlobal > 7){

			dev_model[iGlobal] = (

				dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

				+ dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
				+ dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
				+ dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				+ dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
				+ dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
				+ dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

				+ dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
				+ dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
				+ dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

				+ dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
				+ dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
				+ dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

			);
		}

    }
}

__global__ void setFreeSurfaceToZero(double *dev_c) {

	// Global coordinates for the slowest axis
	long long iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Global y-coordinate
	long long ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Global x-coordinate
	long long iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz;

	if (iyGlobal < dev_ny-FAT){
		dev_c[iGlobal+FAT] = 0.0;
	}
}

__global__ void setFreeSurfaceConditionAdjGpu_3D(double *dev_c) {

	// Global coordinates for the slowest axis
	long long iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Global y-coordinate
	long long ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Global x-coordinate
	long long iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz;

	if (iyGlobal < dev_ny-FAT){

		dev_c[iGlobal+FAT] = 0.0;
		dev_c[iGlobal+2*FAT] = dev_c[iGlobal+2*FAT] - dev_c[iGlobal];
		dev_c[iGlobal+2*FAT-1] = dev_c[iGlobal+2*FAT-1] - dev_c[iGlobal+1];
		dev_c[iGlobal+2*FAT-2] = dev_c[iGlobal+2*FAT-2] - dev_c[iGlobal+2];
		dev_c[iGlobal+2*FAT-3] = dev_c[iGlobal+2*FAT-3] - dev_c[iGlobal+3];
		dev_c[iGlobal] = 0.0;
		dev_c[iGlobal+1] = 0.0;
		dev_c[iGlobal+2] = 0.0;
		dev_c[iGlobal+3] = 0.0;

	}
}

__global__ void subtractNewDebug_3D(double *dev_o, double *dev_n) {

	// Global coordinates for the slowest axis
	long long iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Global y-coordinate
	long long ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Global x-coordinate
	long long iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz;

	if (iyGlobal < dev_ny-FAT){
		for (int iz=0; iz<dev_nz; iz++){
			dev_o[iGlobal+iz] -= dev_n[iGlobal+iz];
		}
	}
}

__global__ void addCurAndStep(double *dev_o, double *dev_c, double *dev_n) {

	// Global coordinates for the slowest axis
	long long iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Global y-coordinate
	long long ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Global x-coordinate
	long long iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz;

	if (iyGlobal < dev_ny-FAT){
		for (int iz=FAT; iz<dev_nz-FAT; iz++){
			dev_n[iGlobal+iz] += 2 * dev_c[iGlobal+iz] - dev_o[iGlobal+iz];
		}
	}
}

__global__ void stepAdjGpuDebug_3D(double *dev_o, double *dev_c, double *dev_n, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1]; // Array for the current wavefield y-slice
    double dev_vel_y[2*FAT+1]; // Array for the scaled velocity y-slice

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction (Remember: each thread has its own version of dev_c_y and dev_vel_y array)
    // Points from the current wavefield time-slice that will be used by the current block
    dev_c_y[1] = dev_c[iGlobalTemp];								dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[2] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[3] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];
	// These ones go to shared memory because used multiple times in Laplacian computation for the z- and x-directions
    shared_c[ixLocal][izLocal] = dev_c[iGlobalTemp+=yStride]; 		shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];
	dev_c_y[5] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[6] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];
	dev_c_y[7] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];
    dev_c_y[8] = dev_c[iGlobalTemp+=yStride];						dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iyGlobal=FAT; iyGlobal<dev_ny-FAT; iyGlobal++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];						dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];						dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];						dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block before updating the value of the shared memory at ixLocal, izLocal
        shared_c[ixLocal][izLocal] = dev_c_y[5]; 		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];						dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];						dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];						dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_c[iGlobalTemp+=yStride];		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_c[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_c[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];
    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_c[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_c[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

        // Apply adjoint stepping operator
        dev_o[iGlobal] = (

            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

        ) + 2.0 * shared_c[ixLocal][izLocal];

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

//
__global__ void derivFwdGpu_3D(double *dev_model, double *dev_data) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_model[iGlobalTemp]; // iy = 0
    dev_c_y[2] = dev_model[iGlobalTemp+=yStride]; // iy = 1
    dev_c_y[3] = dev_model[iGlobalTemp+=yStride]; // iy = 2
    shared_c[ixLocal][izLocal] = dev_model[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
    dev_c_y[5] = dev_model[iGlobalTemp+=yStride]; // iy = 4
    dev_c_y[6] = dev_model[iGlobalTemp+=yStride]; // iy = 5
    dev_c_y[7] = dev_model[iGlobalTemp+=yStride];// iy = 6
    dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5]; // Store the middle one in the shared memory (it will be re-used to compute the Laplacian in the z- and x-directions)
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // The last point of the stencil now points to the next y-slice

        // Remark on assignments just above:
        // iyTemp = iy + FAT
        // This guy points to the iy with the largest y-index needed to compute the Laplacian at the new y-position

        // Load the halos in the x-direction
        // Threads with x-index ranging from 0,...,FAT will load the first and last FAT elements of the block on the x-axis to shared memory
        if (threadIdx.y < FAT) {
			shared_c[threadIdx.y][izLocal] = dev_model[iGlobal-dev_nz*FAT]; // Left side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_model[iGlobal+dev_nz*BLOCK_SIZE_X]; // Right side
    	}

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
    		shared_c[ixLocal][threadIdx.x] = dev_model[iGlobal-FAT]; // Up
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_model[iGlobal+BLOCK_SIZE_Z]; // Down
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads(); // Synchronise all threads within each block
    	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
    	// We can now compute the Laplacian value at each point of the entire block slice

        // Apply forward stepping operator
        dev_data[iGlobal] = (

            dev_coeff[C0] * shared_c[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

__global__ void derivTopAdjGpu_3D(double *dev_model, double *dev_data) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Current wavefield y-slice block


    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1]; // Array for the current wavefield y-slice

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction (Remember: each thread has its own version of dev_c_y and dev_vel_y array)
    // Points from the current wavefield time-slice that will be used by the current block
    dev_c_y[1] = dev_data[iGlobalTemp];
    dev_c_y[2] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[3] = dev_data[iGlobalTemp+=yStride];
	// These ones go to shared memory because used multiple times in Laplacian computation for the z- and x-directions
    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride];
	dev_c_y[5] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[6] = dev_data[iGlobalTemp+=yStride];
	dev_c_y[7] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[8] = dev_data[iGlobalTemp+=yStride];

    // Loop over y
    for (long long iyGlobal=FAT; iyGlobal<dev_ny-FAT; iyGlobal++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block before updating the value of the shared memory at ixLocal, izLocal
        shared_c[ixLocal][izLocal] = dev_c_y[5];
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];

    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];

    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

		if (izGlobal == 4){
			// dev_model[iGlobal] = 0.0;
			// iGlobal = iGlobal + yStride;
		}
		if (izGlobal == 5){
	        dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
	            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

				- dev_coeff[CZ2] * shared_c[ixLocal][izLocal]
				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal+1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal+2]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
	            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

	        );

	        // Move forward one grid point in the y-direction
	        // iGlobal = iGlobal + yStride;
		}

		if (izGlobal == 6){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
	            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5])

				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal-1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
	            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

	        );

			// Move forward one grid point in the y-direction
	        // iGlobal = iGlobal + yStride;

		}
		if (izGlobal == 7){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
	            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5])

				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal-2]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
	            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

	        );
			// Move forward one grid point in the y-direction
	        // iGlobal = iGlobal + yStride;
		}

		if (izGlobal > 7){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
	            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5])

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
	            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

	        );
			// iGlobal = iGlobal + yStride;
		}
		iGlobal = iGlobal + yStride;
    }
}

__global__ void derivBodyAdjGpu_3D(double *dev_model, double *dev_data) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Current wavefield y-slice block


    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + (blockIdx.x + 1) * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1]; // Array for the current wavefield y-slice

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction (Remember: each thread has its own version of dev_c_y and dev_vel_y array)
    // Points from the current wavefield time-slice that will be used by the current block
    dev_c_y[1] = dev_data[iGlobalTemp];
    dev_c_y[2] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[3] = dev_data[iGlobalTemp+=yStride];
	// These ones go to shared memory because used multiple times in Laplacian computation for the z- and x-directions
    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride];
	dev_c_y[5] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[6] = dev_data[iGlobalTemp+=yStride];
	dev_c_y[7] = dev_data[iGlobalTemp+=yStride];
    dev_c_y[8] = dev_data[iGlobalTemp+=yStride];

    // Loop over y
    for (long long iyGlobal=FAT; iyGlobal<dev_ny-FAT; iyGlobal++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block before updating the value of the shared memory at ixLocal, izLocal
        shared_c[ixLocal][izLocal] = dev_c_y[5];
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];

    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];

    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

		// if (izGlobal > 7){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
	            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5])

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
	            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

	        );
			// iGlobal = iGlobal + yStride;
		// }
		iGlobal = iGlobal + yStride;
    }
}

__global__ void derivFwdVelGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_model[iGlobalTemp]; // iy = 0
    dev_c_y[2] = dev_model[iGlobalTemp+=yStride]; // iy = 1
    dev_c_y[3] = dev_model[iGlobalTemp+=yStride]; // iy = 2
    shared_c[ixLocal][izLocal] = dev_model[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
    dev_c_y[5] = dev_model[iGlobalTemp+=yStride]; // iy = 4
    dev_c_y[6] = dev_model[iGlobalTemp+=yStride]; // iy = 5
    dev_c_y[7] = dev_model[iGlobalTemp+=yStride];// iy = 6
    dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5]; // Store the middle one in the shared memory (it will be re-used to compute the Laplacian in the z- and x-directions)
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // The last point of the stencil now points to the next y-slice

        // Remark on assignments just above:
        // iyTemp = iy + FAT
        // This guy points to the iy with the largest y-index needed to compute the Laplacian at the new y-position

        // Load the halos in the x-direction
        // Threads with x-index ranging from 0,...,FAT will load the first and last FAT elements of the block on the x-axis to shared memory
        if (threadIdx.y < FAT) {
			shared_c[threadIdx.y][izLocal] = dev_model[iGlobal-dev_nz*FAT]; // Left side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_model[iGlobal+dev_nz*BLOCK_SIZE_X]; // Right side
    	}

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
    		shared_c[ixLocal][threadIdx.x] = dev_model[iGlobal-FAT]; // Up
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_model[iGlobal+BLOCK_SIZE_Z]; // Down
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads(); // Synchronise all threads within each block
    	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
    	// We can now compute the Laplacian value at each point of the entire block slice

        // Apply forward stepping operator
        dev_data[iGlobal] = dev_vel2Dtw2[iGlobal] * (

            dev_coeff[C0] * shared_c[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

        ) + 2.0 * shared_c[ixLocal][izLocal];

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

__global__ void derivAdjVelGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1]; // Array for the current wavefield y-slice
    double dev_vel_y[2*FAT+1]; // Array for the scaled velocity y-slice

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

	// Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_data[iGlobalTemp]; // iy = 0
	dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[2] = dev_data[iGlobalTemp+=yStride]; // iy = 1
	dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[3] = dev_data[iGlobalTemp+=yStride]; // iy = 2
	dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];

    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
	shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[5] = dev_data[iGlobalTemp+=yStride]; // iy = 4
	dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[6] = dev_data[iGlobalTemp+=yStride]; // iy = 5
	dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[7] = dev_data[iGlobalTemp+=yStride];// iy = 6
	dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[8] = dev_data[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7
	dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iyGlobal=FAT; iyGlobal<dev_ny-FAT; iyGlobal++){

        // Update temporary arrays with current wavefield values along the y-axis
		dev_c_y[0] = dev_c_y[1];
		dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];
		dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];
		dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5];
		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];
		dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];
		dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];
		dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];
		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
			// Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];

    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
			// Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

		if (izGlobal == 4){
			dev_model[iGlobal] = 0.0;
		}
		if (izGlobal == 5){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
	            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ2] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]
				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
	            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

	        ) + 2.0 * shared_c[ixLocal][izLocal];

		}

		if (izGlobal == 6){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
	            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ3] * shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1]
				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
	            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

	        ) + 2.0 * shared_c[ixLocal][izLocal];

		}
		if (izGlobal == 7){

			dev_model[iGlobal] = (

				dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
	            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

				- dev_coeff[CZ4] * shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2]

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
	            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

	        ) + 2.0 * shared_c[ixLocal][izLocal];
		}

		if (izGlobal > 7){

			dev_model[iGlobal] = (

	            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

	            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
	            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
	            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

	            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
	            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
	            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

	            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
	            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
	            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

	            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
	            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
	            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])

	        ) + 2.0 * shared_c[ixLocal][izLocal];
		}
		iGlobal = iGlobal + yStride;
    }
}
