#include "hip/hip_runtime.h"
#include "varDeclare_3D.h"
#include <stdio.h>

/******************************************************************************/
/******************************* Debug Laplacian ******************************/
/******************************************************************************/
//
__global__ void LaplacianFwdGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_model[iGlobalTemp]; // iy = 0
    dev_c_y[2] = dev_model[iGlobalTemp+=yStride]; // iy = 1
    dev_c_y[3] = dev_model[iGlobalTemp+=yStride]; // iy = 2
    shared_c[ixLocal][izLocal] = dev_model[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
    dev_c_y[5] = dev_model[iGlobalTemp+=yStride]; // iy = 4
    dev_c_y[6] = dev_model[iGlobalTemp+=yStride]; // iy = 5
    dev_c_y[7] = dev_model[iGlobalTemp+=yStride];// iy = 6
    dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update values along the y-axis
        dev_c_y[0] = dev_c_y[1];
        dev_c_y[1] = dev_c_y[2];
        dev_c_y[2] = dev_c_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5]; // Store the middle one in the shared memory (it will be re-used to compute the Laplacian in the z- and x-directions)
        dev_c_y[5] = dev_c_y[6];
        dev_c_y[6] = dev_c_y[7];
        dev_c_y[7] = dev_c_y[8];
        dev_c_y[8] = dev_model[iGlobalTemp+=yStride]; // The last point of the stencil now points to the next y-slice

        // Remark on assignments just above:
        // iyTemp = iy + FAT
        // This guy points to the iy with the largest y-index needed to compute the Laplacian at the new y-position

        // Load the halos in the x-direction
        // Threads with x-index ranging from 0,...,FAT will load the first and last FAT elements of the block on the x-axis to shared memory
        if (threadIdx.y < FAT) {
			shared_c[threadIdx.y][izLocal] = dev_model[iGlobal-dev_nz*FAT]; // Left side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_model[iGlobal+dev_nz*BLOCK_SIZE_X]; // Right side
    	}

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
    		shared_c[ixLocal][threadIdx.x] = dev_model[iGlobal-FAT]; // Up
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_model[iGlobal+BLOCK_SIZE_Z]; // Down
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads(); // Synchronise all threads within each block
    	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
    	// We can now compute the Laplacian value at each point of the entire block slice

        // Apply forward stepping operator
        dev_data[iGlobal] = dev_vel2Dtw2[iGlobal] * (

            dev_coeff[C0] * shared_c[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] )
            + dev_coeff[CY1] * ( dev_c_y[3] + dev_c_y[5] )

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] )
            + dev_coeff[CY2] * ( dev_c_y[2] + dev_c_y[6] )

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] + dev_c_y[7] )

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] + dev_c_y[8] )

        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;

    }
}

/* Forward stepper (no damping) */
__global__ void LaplacianAdjGpu_3D(double *dev_model, double *dev_data, double *dev_vel2Dtw2) {

    // Allocate shared memory for a specific block
	__shared__ double shared_c[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT];  // Current wavefield y-slice block
	__shared__ double shared_vel[BLOCK_SIZE_X+2*FAT][BLOCK_SIZE_Z+2*FAT]; // Scaled velocity y-slice block

    // Global coordinates for the faster two axes (z and x)
	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Coordinate of current thread on the z-axis
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Coordinate of current thread on the x-axis

    // Local coordinates for the fastest two axes
	long long izLocal = FAT + threadIdx.x; // z-coordinate on the local grid stored in shared memory
	long long ixLocal = FAT + threadIdx.y; // x-coordinate on the local grid stored in shared memory

    // Allocate (on global memory?) the array that will store the wavefield values in the y-direction
    // Each thread will have its own version of this array
    // Question: is that on the global memory? -> can it fit in the register?
    // Why do we create this temporary array and not call it directly from global memory?
    double dev_c_y[2*FAT+1];
	double dev_vel_y[2*FAT+1];

    // Number of elements in one y-slice
    long long yStride = dev_nz * dev_nx;

    // Global index of the first element at which we are going to compute the Laplacian
    // Skip the first FAT elements on the y-axis
    long long iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal;

    // Global index of the element with the smallest y-position needed to compute Laplacian at iGlobal
    long long iGlobalTemp = iGlobal - FAT * yStride;

    // Load the values along the y-direction into dev_c_y (Remember: each thread has its own version of this array)
    dev_c_y[1] = dev_data[iGlobalTemp]; // iy = 0
	dev_vel_y[1] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[2] = dev_data[iGlobalTemp+=yStride]; // iy = 1
	dev_vel_y[2] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[3] = dev_data[iGlobalTemp+=yStride]; // iy = 2
	dev_vel_y[3] = dev_vel2Dtw2[iGlobalTemp];

    shared_c[ixLocal][izLocal] = dev_data[iGlobalTemp+=yStride]; // Only the central point on the y-axis is stored in the shared memory // iy = 3
	shared_vel[ixLocal][izLocal] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[5] = dev_data[iGlobalTemp+=yStride]; // iy = 4
	dev_vel_y[5] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[6] = dev_data[iGlobalTemp+=yStride]; // iy = 5
	dev_vel_y[6] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[7] = dev_data[iGlobalTemp+=yStride];// iy = 6
	dev_vel_y[7] = dev_vel2Dtw2[iGlobalTemp];

    dev_c_y[8] = dev_data[iGlobalTemp+=yStride]; // At that point, iyTemp = 2*FAT-1 // iy = 7
	dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

    // Loop over y
    for (long long iy=FAT; iy<dev_ny-FAT; iy++){

        // Update temporary arrays with current wavefield values along the y-axis
        dev_c_y[0] = dev_c_y[1];
		dev_vel_y[0] = dev_vel_y[1];
        dev_c_y[1] = dev_c_y[2];
		dev_vel_y[1] = dev_vel_y[2];
        dev_c_y[2] = dev_c_y[3];
		dev_vel_y[2] = dev_vel_y[3];
        dev_c_y[3] = shared_c[ixLocal][izLocal];
		dev_vel_y[3] = shared_vel[ixLocal][izLocal];
		__syncthreads(); // Synchronise all threads within each block
        shared_c[ixLocal][izLocal] = dev_c_y[5];
		shared_vel[ixLocal][izLocal] = dev_vel_y[5]; // Load central points to shared memory (for both current slice and scaled velocity)
        dev_c_y[5] = dev_c_y[6];
		dev_vel_y[5] = dev_vel_y[6];
        dev_c_y[6] = dev_c_y[7];
		dev_vel_y[6] = dev_vel_y[7];
        dev_c_y[7] = dev_c_y[8];
		dev_vel_y[7] = dev_vel_y[8];
        dev_c_y[8] = dev_data[iGlobalTemp+=yStride];
		dev_vel_y[8] = dev_vel2Dtw2[iGlobalTemp];

        // Load the halos in the z-direction
        if (threadIdx.x < FAT) {
            // Top halo
    		shared_c[ixLocal][izLocal-FAT] = dev_data[iGlobal-FAT];
            shared_vel[ixLocal][izLocal-FAT] = dev_vel2Dtw2[iGlobal-FAT];
            // Bottom halo
    		shared_c[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_data[iGlobal+BLOCK_SIZE_Z];
    		shared_vel[ixLocal][izLocal+BLOCK_SIZE_Z] = dev_vel2Dtw2[iGlobal+BLOCK_SIZE_Z];
    	}
        // Load the halos in the x-direction
        if (threadIdx.y < FAT) {
            // Left side
    		shared_c[ixLocal-FAT][izLocal] = dev_data[iGlobal-dev_nz*FAT];
            shared_vel[ixLocal-FAT][izLocal] = dev_vel2Dtw2[iGlobal-dev_nz*FAT];
            // Right side
    		shared_c[ixLocal+BLOCK_SIZE_X][izLocal] = dev_data[iGlobal+dev_nz*BLOCK_SIZE_X];
    		shared_vel[ixLocal+BLOCK_SIZE_X][izLocal] = dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE_X];
    	}

        // Wait until all threads of this block have loaded the slice y-slice into shared memory
        __syncthreads();

        // Apply adjoint stepping operator
        dev_model[iGlobal] = (

            dev_coeff[C0] * shared_c[ixLocal][izLocal] * shared_vel[ixLocal][izLocal]

            + dev_coeff[CZ1] * ( shared_c[ixLocal][izLocal-1] * shared_vel[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] * shared_vel[ixLocal][izLocal+1] )
            + dev_coeff[CX1] * ( shared_c[ixLocal-1][izLocal] * shared_vel[ixLocal-1][izLocal] + shared_c[ixLocal+1][izLocal] * shared_vel[ixLocal+1][izLocal])
            + dev_coeff[CY1] * ( dev_c_y[3] * dev_vel_y[3] + dev_c_y[5] * dev_vel_y[5])

            + dev_coeff[CZ2] * ( shared_c[ixLocal][izLocal-2] * shared_vel[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] * shared_vel[ixLocal][izLocal+2] )
            + dev_coeff[CX2] * ( shared_c[ixLocal-2][izLocal] * shared_vel[ixLocal-2][izLocal] + shared_c[ixLocal+2][izLocal] * shared_vel[ixLocal+2][izLocal])
            + dev_coeff[CY2] * ( dev_c_y[2] * dev_vel_y[2] + dev_c_y[6] * dev_vel_y[6])

            + dev_coeff[CZ3] * ( shared_c[ixLocal][izLocal-3] * shared_vel[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] * shared_vel[ixLocal][izLocal+3] )
            + dev_coeff[CX3] * ( shared_c[ixLocal-3][izLocal] * shared_vel[ixLocal-3][izLocal] + shared_c[ixLocal+3][izLocal] * shared_vel[ixLocal+3][izLocal] )
            + dev_coeff[CY3] * ( dev_c_y[1] * dev_vel_y[1] + dev_c_y[7] * dev_vel_y[7])

            + dev_coeff[CZ4] * ( shared_c[ixLocal][izLocal-4] * shared_vel[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] * shared_vel[ixLocal][izLocal+4] )
            + dev_coeff[CX4] * ( shared_c[ixLocal-4][izLocal] * shared_vel[ixLocal-4][izLocal] + shared_c[ixLocal+4][izLocal] * shared_vel[ixLocal+4][izLocal] )
            + dev_coeff[CY4] * ( dev_c_y[0] * dev_vel_y[0] + dev_c_y[8] * dev_vel_y[8])
        );

        // Move forward one grid point in the y-direction
        iGlobal = iGlobal + yStride;
    }
}

/******************************************************************************/
/******************************** Imaging kernels *****************************/
/******************************************************************************/
// Forward non-extended
__global__ void imagingFwdGpu_3D_zLoop(double *dev_model, double *dev_data, double *dev_sourceWavefieldDts) {

	int iyGlobal = FAT + blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Coordinate on y-axis

	// Make sure you are inside FAT and ny-FAT (non-included)
	if (iyGlobal < dev_ny-FAT){

		int ixGlobal = FAT + blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Coordinate on x-axis
		int iGlobal = iyGlobal * dev_yStride + ixGlobal * dev_nz; // Global coordinate on the time slice
		// Loop over z-axis
		for (int iz=FAT; iz<dev_nz-FAT; iz++){
			dev_data[iGlobal] = dev_model[iGlobal] * dev_sourceWavefieldDts[iGlobal];
			iGlobal=iGlobal+1;
		}
	}
}

__global__ void imagingFwdGpu_3D_yLoop(double *dev_model, double *dev_data, double *dev_sourceWavefieldDts) {

	// Global coordinates for the faster two axes (z and x)
	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE_Z + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE_X + threadIdx.y; // Global x-coordinate
    int yStride = dev_nz * dev_nx;
    int iGlobal = FAT * yStride + dev_nz * ixGlobal + izGlobal; // Global position on the cube

    for (int iy=FAT; iy<dev_ny-FAT; iy++){
		dev_data[iGlobal] = dev_model[iGlobal] * dev_sourceWavefieldDts[iGlobal];
		iGlobal+=yStride;
	}
}
