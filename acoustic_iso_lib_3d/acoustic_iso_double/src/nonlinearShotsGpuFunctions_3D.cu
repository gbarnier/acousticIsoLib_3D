#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>
#include "nonlinearShotsGpuFunctions_3D.h"
#include "varDeclare_3D.h"
#include "kernelsGpu_3D.cu"
#include "cudaErrors_3D.cu"
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <ctime>
#include <stdio.h>
#include <assert.h>

/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
bool getGpuInfo_3D(std::vector<int> gpuList, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << gpuList.size() << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;
		std::cout << "Id of requested GPUs: ";
		for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
			if (iGpu<gpuList.size()-1){std::cout << gpuList[iGpu] << ", ";}
 			else{ std::cout << gpuList[iGpu] << std::endl;}
		}

		// Driver version
		std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl;

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

	// Check that the number of requested GPU is less or equal to the total number of available GPUs
	if (gpuList.size()>nDevice) {
		std::cout << "**** ERROR [getGpuInfo_3D]: Number of requested GPU greater than available GPUs ****" << std::endl;
		return false;
	}

	// Check that the GPU numbers in the list are between 0 and nGpu-1
	for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
		if (gpuList[iGpu]<0 || gpuList[iGpu]>nDevice-1){
			std::cout << "**** ERROR [getGpuInfo_3D]: One of the element of the GPU Id list is not a valid GPU Id number ****" << std::endl;
			return false;
		}
	}

	return true;
}
void initNonlinearGpu_3D(double dz, double dx, double dy, int nz, int nx, int ny, int nts, double dts, int sub, int minPad, int blockSize, double alphaCos, int nGpu, int iGpuId, int iGpuAlloc){

	// Set GPU
	hipSetDevice(iGpuId);

	// Host variables
	host_nz = nz;
	host_nx = nx;
    host_ny = ny;
	host_nModel = nz * nx * ny;
	host_yStride = nz * nx;
	host_nts = nts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;

	/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
	// Only one GPU will perform the following
	if (iGpuId == iGpuAlloc) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		// Data and model
		dev_modelRegDtw = new double*[nGpu];
		dev_dataRegDts = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new long long*[nGpu];
		dev_receiversPositionReg = new long long*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

		// Debug model and data
		dev_modelDebug = new double*[nGpu];
		dev_dataDebug = new double*[nGpu];

		// Compute and transfer stream
		topStream = new hipStream_t[nGpu];
		compStream = new hipStream_t[nGpu];

	}

	/**************************** COMPUTE LAPLACIAN COEFFICIENTS ************************/
	// Compute coefficients for 8th order central finite difference Laplacian
	double host_coeff[COEFF_SIZE] = get_coeffs((double)dz,(double)dx,(double)dy); // Stored on host

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	// Time interpolation filter length / half length
	int hInterpFilter = host_sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Subsampling parameter for time interpolation is too high ****" << std::endl;
		assert (1==2);
	}

	// Allocate and fill time interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Padding value is too high ****" << std::endl;
		assert (1==2);
	}
	double cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		double arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Blocksize value from parfile does not match value from varDeclare.h file ****" << std::endl;
		assert (1==2);
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_coeff), host_coeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nTimeInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hTimeInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_timeInterpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(double), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ny), &ny, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_yStride), &host_yStride, sizeof(long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nModel), &host_nModel, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

}
void allocateNonlinearGpu_3D(double *vel2Dtw2, int iGpu, int iGpuId){

	// Get GPU number
	hipSetDevice(iGpuId);

	// Scaled velocity
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nModel*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nModel*sizeof(double), hipMemcpyHostToDevice));

	// Allocate time slices on device
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nModel*sizeof(double))); // Allocate time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nModel*sizeof(double)));

}
void deallocateNonlinearGpu_3D(int iGpu, int iGpuId){
	hipSetDevice(iGpuId); // Set device number on GPU cluster
	cuda_call(hipFree(dev_vel2Dtw2[iGpu])); // Deallocate scaled velocity
	cuda_call(hipFree(dev_p0[iGpu]));
	cuda_call(hipFree(dev_p1[iGpu]));
}

/****************************************************************************************/
/******************************* Nonlinear forward propagation **************************/
/****************************************************************************************/
void propShotsFwdGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	// std::cout << "No free surface forward" << std::endl;
	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){
		// if (iGpuId == 1){
			// std::cout << "iGpuId = " << iGpuId << " its = " << its << std::endl;
		// }
		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsFwdFreeSurfaceGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	// std::cout << "Free surface forward" << std::endl;
	// printf("Free surface");
	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){
		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Apply free surface conditions for Laplacian
			kernel_exec(setFreeSurfaceConditionFwdGpu_3D<<<dimGridFreeSurface, dimBlockFreeSurface>>>(dev_p1[iGpu]));

			// Step forward in time
			kernel_exec(stepFwdGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock>>>(dev_p1[iGpu], dev_p0[iGpu]));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

/****************************************************************************************/
/******************************* Nonlinear adjoint propagation **************************/
/****************************************************************************************/
void propShotsAdjGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&topStream[iGpu]);
	hipStreamCreate(&compStream[iGpu]);

	// Create to synchornize top/body for free surface computation
	hipEventCreate(&eventTopFreeSurface);
	hipEventCreate(&eventBodyFreeSurface);
	hipEventCreate(&compStreamDone);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimGridTop(1, nblocky);
	dim3 dimGridBody(nblockx-1, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward in time
			kernel_exec(stepAdjGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdge_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsAdjFreeSurfaceGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&topStream[iGpu]);
	hipStreamCreate(&compStream[iGpu]);

	// Create to synchornize top/body for free surface computation
	hipEventCreate(&eventTopFreeSurface);
	hipEventCreate(&eventBodyFreeSurface);
	hipEventCreate(&compStreamDone);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimGridTop(1, nblocky);
	dim3 dimGridBody(nblockx-1, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// std::cout << "Free surface" << std::endl;
	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step backward in time
			kernel_exec(stepAdjFreeSurfaceGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// std::cout << "Free surface + top-body separation" << std::endl;
	// // Loop over coarse time samples
	// for (int its = host_nts-2; its > -1; its--){
	//
	// 	// Loop over sub loop
	// 	for (int it2 = host_sub-1; it2 > -1; it2--){
	//
	// 		// Compute fine time-step index
	// 		int itw = its * host_sub + it2;
	//
	// 		// Launch top free surface compuation
	// 		stepAdjFreeSurfaceGpu_3D<<<dimGridTop, dimBlock, 0, topStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
	// 		hipEventRecord(eventTopFreeSurface, topStream[iGpu]);
	//
	// 		stepAdjBodyFreeSurfaceGpu_3D<<<dimGridBody, dimBlock, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
	// 		hipStreamWaitEvent(compStream[iGpu], eventTopFreeSurface, 0);
	//
	// 		// Inject data
	// 		interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA, 0, compStream[iGpu]>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]);
	//
	// 		// Damp wavefield
	// 		dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu]);
	//
	// 		// Extract model
	// 		recordSource_3D<<<1, nSourcesReg, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]);
	//
	// 		dev_temp1[iGpu] = dev_p0[iGpu];
	// 		dev_p0[iGpu] = dev_p1[iGpu];
	// 		dev_p1[iGpu] = dev_temp1[iGpu];
	// 		dev_temp1[iGpu] = NULL;
	//
	// 		hipEventRecord(compStreamDone, compStream[iGpu]);
	// 		hipStreamWaitEvent(topStream[iGpu], compStreamDone, 0);
	// 	}
	// }

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}








		// Create two arrays (one on device, other one host)
		// int *host_array, *dev_array;
		// host_array = new int[12];
		// cuda_call(hipMalloc((void**) &dev_array, 12*sizeof(int)));
		// for (int j=0; j<12; j++){
		// 	host_array[j]=j;
		// }
		// cuda_call(hipMemcpy(dev_array, host_array, 12*sizeof(int), hipMemcpyHostToDevice));
		//
		// // My thaang
		// for (int j=0; j<2; j++){
		//
		// 	kA1<<<1, 1, 0, compStream[iGpu]>>>(j, dev_array);
		//
		// 	for (int i=2; i<12; i++){
		// 		kA2<<<1, 1, 0, topStream[iGpu]>>>(i, j, dev_array);
		// 	}
		// 	hipEventRecord(test, NULL);
		// 	hipStreamWaitEvent(compStream[iGpu], test, 0);
		// 	kA3<<<1, 1, 0, compStream[iGpu]>>>(j, dev_array);
		// 	std::cout << "Done j = " << j << std::endl;
		// }
