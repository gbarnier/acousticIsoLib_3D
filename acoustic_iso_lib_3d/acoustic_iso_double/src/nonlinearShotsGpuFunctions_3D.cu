#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>
#include "nonlinearShotsGpuFunctions_3D.h"
#include "varDeclare_3D.h"
#include "kernelsGpu_3D.cu"
#include "cudaErrors_3D.cu"
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <ctime>
#include <stdio.h>
#include <assert.h>

/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
// GPU info
bool getGpuInfo_3D(std::vector<int> gpuList, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << gpuList.size() << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;
		std::cout << "Id of requested GPUs: ";
		for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
			if (iGpu<gpuList.size()-1){std::cout << gpuList[iGpu] << ", ";}
 			else{ std::cout << gpuList[iGpu] << std::endl;}
		}

		// Driver version
		std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl;

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

	// Check that the number of requested GPU is less or equal to the total number of available GPUs
	if (gpuList.size()>nDevice) {
		std::cout << "**** ERROR [getGpuInfo_3D]: Number of requested GPU greater than available GPUs ****" << std::endl;
		return false;
	}

	// Check that the GPU numbers in the list are between 0 and nGpu-1
	for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
		if (gpuList[iGpu]<0 || gpuList[iGpu]>nDevice-1){
			std::cout << "**** ERROR [getGpuInfo_3D]: One of the element of the GPU Id list is not a valid GPU Id number ****" << std::endl;
			return false;
		}
	}

	return true;
}

// Init normal
void initNonlinearGpu_3D(double dz, double dx, double dy, int nz, int nx, int ny, int nts, double dts, int sub, int minPad, int blockSize, double alphaCos, int nGpu, int iGpuId, int iGpuAlloc){

	// Set GPU
	hipSetDevice(iGpuId);

	// Host variables
	host_nz = nz;
	host_nx = nx;
    host_ny = ny;
	host_nModel = nz * nx * ny;
	host_yStride = nz * nx;
	host_nts = nts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;
	host_minPad = minPad;

	/************************* ALLOCATE ARRAYS OF ARRAYS **********************/
	// Only one GPU will perform the following
	if (iGpuId == iGpuAlloc) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		// Data and model
		dev_modelRegDtw = new double*[nGpu];
		dev_dataRegDts = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new long long*[nGpu];
		dev_receiversPositionReg = new long long*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

		// Damping slice
		dev_dampingSlice = new double*[nGpu];

		// Debug model and data
		dev_modelDebug = new double*[nGpu];
		dev_dataDebug = new double*[nGpu];

		// Compute and transfer stream
		topStream = new hipStream_t[nGpu];
		compStream = new hipStream_t[nGpu];

	}

	/********************* COMPUTE LAPLACIAN COEFFICIENTS *********************/
	// Compute coefficients for 8th order central finite difference Laplacian
	double host_coeff[COEFF_SIZE] = get_coeffs((double)dz,(double)dx,(double)dy); // Stored on host

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	// Time interpolation filter length / half length
	int hInterpFilter = host_sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Subsampling parameter for time interpolation is too high ****" << std::endl;
		throw std::runtime_error("");
	}

	// Allocate and fill time interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Padding value is too high ****" << std::endl;
		throw std::runtime_error("");
	}
	double cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		double arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
		// std::cout << "Damp array gpu [" << iFilter-FAT << "] = " << arg << std::endl;
	}

	// Check that the block size is consistent between parfile and "varDeclare_3D.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Blocksize value from parfile does not match value from varDeclare.h file ****" << std::endl;
		throw std::runtime_error("");
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_coeff), host_coeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nTimeInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hTimeInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_timeInterpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(double), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ny), &ny, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_yStride), &host_yStride, sizeof(long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nModel), &host_nModel, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

}

// Allocate normal
void allocateNonlinearGpu_3D(double *vel2Dtw2, int iGpu, int iGpuId){

	// Get GPU number
	hipSetDevice(iGpuId);

	// Scaled velocity
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nModel*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nModel*sizeof(double), hipMemcpyHostToDevice));

	// Damping slice
	// cuda_call(hipMalloc((void**) &dev_dampingSlice[iGpu], host_nModel*sizeof(double))); // Allocate scaled velocity model on device
	// cuda_call(hipMemcpy(dev_dampingSlice[iGpu], vel2Dtw2, host_nModel*sizeof(double), hipMemcpyHostToDevice));

	// Allocate time slices on device
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nModel*sizeof(double))); // Allocate time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nModel*sizeof(double)));
}

// Init Ginsu
void initNonlinearGinsuGpu_3D(double dz, double dx, double dy, int nts, double dts, int sub, int blockSize, double alphaCos, int nGpu, int iGpuId, int iGpuAlloc){

	// Set GPU
	hipSetDevice(iGpuId);

	// Host variables
	host_nts = nts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;

	/************************ ALLOCATE ARRAYS OF ARRAYS ***********************/
	// Only one GPU will perform the following
	if (iGpuId == iGpuAlloc) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		// Data and model
		dev_modelRegDtw = new double*[nGpu];
		dev_dataRegDts = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new long long*[nGpu];
		dev_receiversPositionReg = new long long*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

		// Damping slice
		dev_dampingSlice = new double*[nGpu];

		// Debug model and data
		dev_modelDebug = new double*[nGpu];
		dev_dataDebug = new double*[nGpu];

		// Compute and transfer stream
		topStream = new hipStream_t[nGpu];
		compStream = new hipStream_t[nGpu];

	}

	/*********************** COMPUTE LAPLACIAN COEFFICIENTS *******************/
	// Compute coefficients for 8th order central finite difference Laplacian
	double host_coeff[COEFF_SIZE] = get_coeffs((double)dz,(double)dx,(double)dy); // Stored on host

	/********************** COMPUTE TIME-INTERPOLATION FILTER *****************/
	// Time interpolation filter length / half length
	int hInterpFilter = host_sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (nGpu>N_GPU_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Number of requested GPUs exceeds number allowed for constant memory storage ****" << std::endl;
		throw std::runtime_error("");
	}

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Subsampling parameter for time interpolation is too high ****" << std::endl;
		throw std::runtime_error("");
	}

	// Allocate and fill time interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	// Check that the block size is consistent between parfile and "varDeclare_3D.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Blocksize value from parfile does not match value from varDeclare.h file ****" << std::endl;
		throw std::runtime_error("");
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_coeff), host_coeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nTimeInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hTimeInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_timeInterpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula

	// Time parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

}

// Allocate Ginsu
void allocateSetNonlinearGinsuGpu_3D(int nz, int nx, int ny, int minPad, int blockSize, double alphaCos, double *vel2Dtw2, int iGpu, int iGpuId){

	// Set GPU
	hipSetDevice(iGpuId);

	// Host variables
	host_nz_ginsu[iGpu] = nz;
	host_nx_ginsu[iGpu] = nx;
    host_ny_ginsu[iGpu] = ny;
	host_nModel_ginsu[iGpu] = nz * nx * ny;
	host_yStride_ginsu[iGpu] = nz * nx;
	host_minPad_ginsu[iGpu] = minPad;

	/******************** COMPUTE COSINE DAMPING COEFFICIENTS *****************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Padding value is too high ****" << std::endl;
		throw std::runtime_error("");
	}

	// Allocate array to store damping coefficients on host
	double host_cosDampingCoeffGinsuTemp[host_minPad_ginsu[iGpu]];

	// Compute array coefficients
	for (int iFilter=FAT; iFilter<FAT+host_minPad_ginsu[iGpu]; iFilter++){
		double arg = M_PI / (1.0 * host_minPad_ginsu[iGpu]) * 1.0 * (host_minPad_ginsu[iGpu]-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		host_cosDampingCoeffGinsuTemp[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare_3D.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR [nonlinearShotsGpuFunctions_3D]: Blocksize value from parfile does not match value from varDeclare.h file ****" << std::endl;
		throw std::runtime_error("");
	}

	/********************** COPY TO CONSTANT MEMORY ***************************/
	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeffGinsuConstant), &host_cosDampingCoeffGinsuTemp, host_minPad_ginsu[iGpu]*sizeof(double), iGpu*PAD_MAX*sizeof(double), hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad_ginsu), &host_minPad_ginsu[iGpu], sizeof(int), iGpu*sizeof(int), hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz_ginsu), &nz, sizeof(int), iGpu*sizeof(int), hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx_ginsu), &nx, sizeof(int), iGpu*sizeof(int), hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ny_ginsu), &ny, sizeof(int), iGpu*sizeof(int), hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_yStride_ginsu), &host_yStride_ginsu[iGpu], sizeof(long long), iGpu*sizeof(long long), hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nModel_ginsu), &host_nModel_ginsu[iGpu], sizeof(unsigned long long), iGpu*sizeof(unsigned long long), hipMemcpyHostToDevice));

	// Copy to global memory
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nModel_ginsu[iGpu]*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nModel_ginsu[iGpu]*sizeof(double), hipMemcpyHostToDevice));

	// Allocate time slices on device
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nModel_ginsu[iGpu]*sizeof(double))); // Allocate time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nModel_ginsu[iGpu]*sizeof(double)));
}

// Deallocate
void deallocateNonlinearGpu_3D(int iGpu, int iGpuId){
	hipSetDevice(iGpuId); // Set device number on GPU cluster
	cuda_call(hipFree(dev_vel2Dtw2[iGpu])); // Deallocate scaled velocity
	cuda_call(hipFree(dev_p0[iGpu]));
	cuda_call(hipFree(dev_p1[iGpu]));
}

/****************************************************************************************/
/******************************* Nonlinear forward propagation **************************/
/****************************************************************************************/
void propShotsFwdGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));
	// cuda_call(hipMemset(dev_dampingSlice[iGpu], 0, host_nModel*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocksize = 32
	int nblockx32 = (host_nz-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){

		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu]));
			// kernel_exec(dampCosineEdge_3D_32<<<dimGridTest1, dimBlockTest1>>>(dev_p1[iGpu], dev_p0[iGpu]));
			// kernel_exec(dampCosineEdge_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));
			// kernel_exec(dampCosineEdge_3D_8<<<dimGridTest2, dimBlockTest2>>>(dev_p0[iGpu], dev_p1[iGpu]));
			// kernel_exec(dampCosineEdge_3DBenchmark<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_dampingSlice[iGpu]));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsFwdGinsuGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_X;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocksize = 32
	int nblockx32 = (host_nz_ginsu[iGpu]-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx_ginsu[iGpu]-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){

		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGinsuGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu], iGpu));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdgeGinsu_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu], iGpu));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsFwdGpu_3D_dampTest(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId, double *dampVolume) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));
	// cuda_call(hipMemset(dev_dampingSlice[iGpu], 0, host_nModel*sizeof(double)));

	// double *p0, *p1, *pDiff;
	// p0 = new double[host_nModel];
	// p1 = new double[host_nModel];
	// pDiff = new double[host_nModel];
	// std::memset(p0, 0, host_nModel*sizeof(double));
	// std::memset(p1, 0, host_nModel*sizeof(double));

	// Set p0 and p1 to zero
	// std::fill(p0, p0+host_nModel, 0.0);
	// std::fill(p1, p1+host_nModel, 0.0);
	//
	// std::cout << "p0 min = " << *std::min_element(p0, p0+host_nModel) << std::endl;
	// std::cout << "p0 max = " << *std::max_element(p0, p0+host_nModel) << std::endl;
	// std::cout << "p1 min = " << *std::min_element(p1, p1+host_nModel) << std::endl;
	// std::cout << "p1 max = " << *std::max_element(p1, p1+host_nModel) << std::endl;

	// Fill in p0 and p1 with ones
	// for (long long iy = FAT; iy < host_ny-FAT; iy++){
	// 	for (long long ix = FAT; ix < host_nx-FAT; ix++){
	// 		for (long long iz = FAT; iz < host_nz-FAT; iz++){
	// 			long long iGlobal = iy * host_nx * host_nz + ix * host_nz + iz;
	// 			p0[iGlobal] = 1.0;
	// 			p1[iGlobal] = 1.0;
	// 		}
	// 	}
	// }

	// cuda_call(hipMemcpy(dev_p0[iGpu], p0, host_nModel*sizeof(double), hipMemcpyHostToDevice));
	// cuda_call(hipMemcpy(dev_p1[iGpu], p1, host_nModel*sizeof(double), hipMemcpyHostToDevice));
	//
	// std::cout << "p0 min after = " << *std::min_element(p0,p0+host_nModel) << std::endl;
	// std::cout << "p0 max after = " << *std::max_element(p0,p0+host_nModel) << std::endl;
	// std::cout << "p1 min after = " << *std::min_element(p1,p1+host_nModel) << std::endl;
	// std::cout << "p1 max after = " << *std::max_element(p1,p1+host_nModel) << std::endl;

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Damping kernels for front / back
	int nBlockDamp = (host_minPad+BLOCK_SIZE_DAMP-1) / BLOCK_SIZE_DAMP;
	dim3 dimGridDampFront(nblocky, nBlockDamp);
	dim3 dimBlockDampFront(BLOCK_SIZE_X, BLOCK_SIZE_DAMP);
	std::cout << "host_minPad = " << host_minPad << std::endl;
	std::cout << "nBlockDamp = " << nBlockDamp << std::endl;
	std::cout << "BLOCK_SIZE_DAMP = " << BLOCK_SIZE_DAMP << std::endl;

	// Damping kernels for left / right
	dim3 dimGridDampLeft(nblockz, nBlockDamp);
	dim3 dimBlockDampLeft(BLOCK_SIZE_Y, BLOCK_SIZE_DAMP);

	// Damping kernels for top / bottom
	dim3 dimGridDampTop(nblocky, nblockz);
	dim3 dimBlockDampTop(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// Damp front / back
	// kernel_exec(dampFrontBack_3D<<<dimGridDampFront, dimBlockDampFront>>>(dev_p0[iGpu], dev_p1[iGpu]));
	//
	// // Damp left / right
	// kernel_exec(dampLeftRight_3D<<<dimGridDampLeft, dimBlockDampLeft>>>(dev_p0[iGpu], dev_p1[iGpu]));
	//
	// // // Damp top / bottom
	// kernel_exec(dampTopBottom_3D<<<dimGridDampTop, dimBlockDampTop>>>(dev_p0[iGpu], dev_p1[iGpu]));


	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){

		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp front / back
			kernel_exec(dampFrontBack_3D<<<dimGridDampFront, dimBlockDampFront>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Damp left / right
			kernel_exec(dampLeftRight_3D<<<dimGridDampLeft, dimBlockDampLeft>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Damp top / bottom
			// kernel_exec(dampTopBottom_3D<<<dimGridDampTop, dimBlockDampTop>>>(dev_p0[iGpu], dev_p1[iGpu]));
			//
			// // Extract and interpolate data
			// kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	// Compare damp volumes CPU/GPU
	// cuda_call(hipMemcpy(p0, dev_p0[iGpu], host_nModel*sizeof(double), hipMemcpyDeviceToHost));
	// cuda_call(hipMemcpy(p1, dev_p1[iGpu], host_nModel*sizeof(double), hipMemcpyDeviceToHost));
	//
	// for (long long iGlobal = 0; iGlobal < host_nModel; iGlobal++){
	//
	// 	pDiff[iGlobal] = p0[iGlobal] - dampVolume[iGlobal];
	//
	// }
	//
	// std::cout << "pDiff min after = " << *std::min_element(pDiff,pDiff+host_nModel) << std::endl;
	// std::cout << "pDiff max after = " << *std::max_element(pDiff,pDiff+host_nModel) << std::endl;

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsFwdFreeSurfaceGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Blocksize = 32
	int nblockx32 = (host_nz-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){
		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Apply free surface conditions for Laplacian
			kernel_exec(setFreeSurfaceConditionFwdGpu_3D<<<dimGridFreeSurface, dimBlockFreeSurface>>>(dev_p1[iGpu]));

			// Step forward in time
			kernel_exec(stepFwdGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			// kernel_exec(dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock>>>(dev_p1[iGpu], dev_p0[iGpu]));
			kernel_exec(dampCosineEdgeFreeSurface_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsFwdFreeSurfaceGinsuGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));

	// Laplacian grid and blocks
	int nblockx = (host_nz_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny_ginsu[iGpu]-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Blocksize = 32
	int nblockx32 = (host_nz_ginsu[iGpu]-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx_ginsu[iGpu]-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	// Loop over coarse time samples
	for (long long its = 0; its < host_nts-1; its++){
		// Loop over sub loop
		for (long long it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			long long itw = its * host_sub + it2;

			// Apply free surface conditions for Laplacian
			kernel_exec(setFreeSurfaceConditionFwdGinsuGpu_3D<<<dimGridFreeSurface, dimBlockFreeSurface>>>(dev_p1[iGpu], iGpu));

			// Step forward in time
			kernel_exec(stepFwdGinsuGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu], iGpu));

			// Inject source
			kernel_exec(injectSourceLinear_3D<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			// kernel_exec(dampCosineEdgeFreeSurfaceGinsu_3D<<<dimGrid, dimBlock>>>(dev_p1[iGpu], dev_p0[iGpu], iGpu));
			kernel_exec(dampCosineEdgeFreeSurfaceGinsu_32_3D<<<dimGrid32, dimBlock32>>>(dev_p1[iGpu], dev_p0[iGpu], iGpu));

			// Extract and interpolate data
			kernel_exec(recordLinearInterpData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

/****************************************************************************************/
/******************************* Nonlinear adjoint propagation **************************/
/****************************************************************************************/
void propShotsAdjGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocksize = 32
	int nblockx32 = (host_nz-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward in time
			kernel_exec(stepAdjGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			// kernel_exec(dampCosineEdge_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));
			kernel_exec(dampCosineEdge_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsAdjGinsuGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny_ginsu[iGpu]-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);

	// Blocksize = 32
	int nblockx32 = (host_nz_ginsu[iGpu]-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx_ginsu[iGpu]-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward in time
			kernel_exec(stepAdjGinsuGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu], iGpu));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdgeGinsu_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu], iGpu));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsAdjFreeSurfaceGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&topStream[iGpu]);
	hipStreamCreate(&compStream[iGpu]);

	// Create to synchornize top/body for free surface computation
	hipEventCreate(&eventTopFreeSurface);
	hipEventCreate(&eventBodyFreeSurface);
	hipEventCreate(&compStreamDone);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimGridTop(1, nblocky);
	dim3 dimGridBody(nblockx-1, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Blocksize = 32
	int nblockx32 = (host_nz-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	std::clock_t start;
	double duration;
	start = std::clock();

	// std::cout << "Free surface" << std::endl;
	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step backward in time
			kernel_exec(stepAdjFreeSurfaceGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			// kernel_exec(dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));
			kernel_exec(dampCosineEdgeFreeSurface_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	duration = (std::clock() - start) / (double) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// std::cout << "Free surface + top-body separation" << std::endl;
	// // Loop over coarse time samples
	// for (int its = host_nts-2; its > -1; its--){
	//
	// 	// Loop over sub loop
	// 	for (int it2 = host_sub-1; it2 > -1; it2--){
	//
	// 		// Compute fine time-step index
	// 		int itw = its * host_sub + it2;
	//
	// 		// Launch top free surface compuation
	// 		stepAdjFreeSurfaceGpu_3D<<<dimGridTop, dimBlock, 0, topStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
	// 		hipEventRecord(eventTopFreeSurface, topStream[iGpu]);
	//
	// 		stepAdjBodyFreeSurfaceGpu_3D<<<dimGridBody, dimBlock, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]);
	// 		hipStreamWaitEvent(compStream[iGpu], eventTopFreeSurface, 0);
	//
	// 		// Inject data
	// 		interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA, 0, compStream[iGpu]>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]);
	//
	// 		// Damp wavefield
	// 		dampCosineEdgeFreeSurface_3D<<<dimGrid, dimBlock, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_p1[iGpu]);
	//
	// 		// Extract model
	// 		recordSource_3D<<<1, nSourcesReg, 0, compStream[iGpu]>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]);
	//
	// 		dev_temp1[iGpu] = dev_p0[iGpu];
	// 		dev_p0[iGpu] = dev_p1[iGpu];
	// 		dev_p1[iGpu] = dev_temp1[iGpu];
	// 		dev_temp1[iGpu] = NULL;
	//
	// 		hipEventRecord(compStreamDone, compStream[iGpu]);
	// 		hipStreamWaitEvent(topStream[iGpu], compStreamDone, 0);
	// 	}
	// }

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}

void propShotsAdjFreeSurfaceGinsuGpu_3D(double *modelRegDtw, double *dataRegDts, long long *sourcesPositionReg, int nSourcesReg, long long *receiversPositionReg, int nReceiversReg, int iGpu, int iGpuId) {

	// Set device number on GPU cluster
	hipSetDevice(iGpuId);

	// Create streams
	hipStreamCreate(&topStream[iGpu]);
	hipStreamCreate(&compStream[iGpu]);

	// Create to synchornize top/body for free surface computation
	hipEventCreate(&eventTopFreeSurface);
	hipEventCreate(&eventBodyFreeSurface);
	hipEventCreate(&compStreamDone);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(long long), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(long long)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(long long), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nModel_ginsu[iGpu]*sizeof(double)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_Z;
	int nblocky = (host_nx_ginsu[iGpu]-2*FAT) / BLOCK_SIZE_X;
	int nblockz = (host_ny_ginsu[iGpu]-2*FAT+BLOCK_SIZE_Y-1) / BLOCK_SIZE_Y;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimGridTop(1, nblocky);
	dim3 dimGridBody(nblockx-1, nblocky);
	dim3 dimBlock(BLOCK_SIZE_Z, BLOCK_SIZE_X);
	dim3 dimGridFreeSurface(nblocky, nblockz);
	dim3 dimBlockFreeSurface(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	// Blocksize = 32
	int nblockx32 = (host_nz_ginsu[iGpu]-2*FAT+32-1) / 32;
	int nblocky32 = (host_nx_ginsu[iGpu]-2*FAT+32-1) / 32;
	dim3 dimGrid32(nblockx32, nblocky32);
	dim3 dimBlock32(32, 32);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Loop over coarse time samples
	for (int its = host_nts-2; its > -1; its--){
		// Loop over sub loop
		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step backward in time
			kernel_exec(stepAdjFreeSurfaceGinsuGpu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu], iGpu));

			// Inject data
			kernel_exec(interpLinearInjectData_3D<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdgeFreeSurfaceGinsu_3D<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], iGpu));
			// kernel_exec(dampCosineEdgeFreeSurfaceGinsu_32_3D<<<dimGrid32, dimBlock32>>>(dev_p0[iGpu], dev_p1[iGpu], iGpu));

			// Extract model
			kernel_exec(recordSource_3D<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(double), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}
